
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[256];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 16; ++i_c_outer_inner_init) {
    T_matmul_NN_local[(i_c_outer_inner_init * 2)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 32)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 64)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 96)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 1)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 33)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 65)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 2) + 97)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + ((((((int)blockIdx.x) >> 4) * 65536) + (((int)threadIdx.x) * 1024)) + (k_outer_outer * 4)));
    }
    *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + ((((k_outer_outer * 16384) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 4)));
    *(float4*)(kernel_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(kernel + (((((k_outer_outer * 16384) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 8192));
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 16; ++i_c_outer_inner) {
      T_matmul_NN_local[(i_c_outer_inner * 2)] = (T_matmul_NN_local[(i_c_outer_inner * 2)] + (data_shared[(((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4))] * kernel_shared[((((int)threadIdx.x) & 63) * 2)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] + (data_shared[(((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4))] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 128)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 128)] * kernel_shared[((((int)threadIdx.x) & 63) * 2)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 128)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 128)]));
      T_matmul_NN_local[(i_c_outer_inner * 2)] = (T_matmul_NN_local[(i_c_outer_inner * 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 256)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 384)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 129)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 256)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 129)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 384)]));
      T_matmul_NN_local[(i_c_outer_inner * 2)] = (T_matmul_NN_local[(i_c_outer_inner * 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 512)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 640)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 130)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 512)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 130)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 640)]));
      T_matmul_NN_local[(i_c_outer_inner * 2)] = (T_matmul_NN_local[(i_c_outer_inner * 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 768)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 32)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 896)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 64)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 131)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 768)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 96)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 131)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 896)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] + (data_shared[(((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4))] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 1)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] + (data_shared[(((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4))] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 129)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 128)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 1)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 128)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 129)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 257)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 385)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 129)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 257)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 129)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 385)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 513)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 641)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 130)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 513)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 130)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 641)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 769)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 33)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 897)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 65)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 131)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 769)]));
      T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + 97)] + (data_shared[((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 4)) + 131)] * kernel_shared[(((((int)threadIdx.x) & 63) * 2) + 897)]));
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131200)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 96)];
    }
  }
}

