#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "nvmlPower.hpp"

#define M 1024
#define N 4096
#define K 1024

__global__ void warm_up_gpu(long seed, float* __restrict__ temp){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    float it = 0.0f;
    for(int i = 0; i < 5000000; i++)
        it += idx * (hiprand_uniform_double(&state) - 0.5);
    temp[0] = it;
}

int main() {
    float randMax = 1000.0;
    srand((unsigned)time(NULL));
    
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 
    float *d_temp;

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * M * K);
    b   = (float*)malloc(sizeof(float) * N * K);
    out = (float*)malloc(sizeof(float) * M * N);

    // Initialize host arrays
    for(int i = 0; i < M * K; i++) {
        a[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }
    for(int i = 0; i < N * K; i++) {
        b[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a,   sizeof(float) * M * K);
    hipMalloc((void**)&d_b,   sizeof(float) * N * K);
    hipMalloc((void**)&d_out, sizeof(float) * M * N);
    hipMalloc((void**)&d_temp, sizeof(float) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * K, hipMemcpyHostToDevice);

    // Executing kernel 
    warm_up_gpu<<<dim3(256,1,1), dim3(256,1,1)>>>(rand(), d_temp);
    hipDeviceSynchronize();
    nvmlAPIRun();
    for(int i = 0; i < REPEAT; i++) {
        mymatmul_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    }
    hipDeviceSynchronize();
    nvmlAPIEnd();
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N * M, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
