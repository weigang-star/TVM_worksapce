
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    T_matmul_NN_local[(i_c_outer_inner_init * 4)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 16)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 32)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 48)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 64)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 80)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 96)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 112)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 1)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 17)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 33)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 49)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 65)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 81)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 97)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 113)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 2)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 18)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 34)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 50)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 66)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 82)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 98)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 114)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 3)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 19)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 35)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 51)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 67)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 83)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 99)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 4) + 115)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(data + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 16384));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(data + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 32768));
    *(float4*)(data_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(data + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 49152));
    kernel_shared[((int)threadIdx.x)] = kernel[(((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 128)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 4224)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 8320)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 12416)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 16512)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 20480)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 20608)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 24704)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 28800)];
    kernel_shared[(((int)threadIdx.x) + 2048)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 2176)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 32896)];
    kernel_shared[(((int)threadIdx.x) + 2304)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 36864)];
    kernel_shared[(((int)threadIdx.x) + 2432)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 36992)];
    kernel_shared[(((int)threadIdx.x) + 2560)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 2688)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 41088)];
    kernel_shared[(((int)threadIdx.x) + 2816)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 45056)];
    kernel_shared[(((int)threadIdx.x) + 2944)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 45184)];
    kernel_shared[(((int)threadIdx.x) + 3072)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 3200)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 49280)];
    kernel_shared[(((int)threadIdx.x) + 3328)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 53248)];
    kernel_shared[(((int)threadIdx.x) + 3456)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 53376)];
    kernel_shared[(((int)threadIdx.x) + 3584)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 3712)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 57472)];
    kernel_shared[(((int)threadIdx.x) + 3840)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 61440)];
    kernel_shared[(((int)threadIdx.x) + 3968)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 61568)];
    kernel_shared[(((int)threadIdx.x) + 4096)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 65536)];
    kernel_shared[(((int)threadIdx.x) + 4224)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 65664)];
    kernel_shared[(((int)threadIdx.x) + 4352)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 69632)];
    kernel_shared[(((int)threadIdx.x) + 4480)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 69760)];
    kernel_shared[(((int)threadIdx.x) + 4608)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 73728)];
    kernel_shared[(((int)threadIdx.x) + 4736)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 73856)];
    kernel_shared[(((int)threadIdx.x) + 4864)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 77824)];
    kernel_shared[(((int)threadIdx.x) + 4992)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 77952)];
    kernel_shared[(((int)threadIdx.x) + 5120)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 81920)];
    kernel_shared[(((int)threadIdx.x) + 5248)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 82048)];
    kernel_shared[(((int)threadIdx.x) + 5376)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 86016)];
    kernel_shared[(((int)threadIdx.x) + 5504)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 86144)];
    kernel_shared[(((int)threadIdx.x) + 5632)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 90112)];
    kernel_shared[(((int)threadIdx.x) + 5760)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 90240)];
    kernel_shared[(((int)threadIdx.x) + 5888)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 94208)];
    kernel_shared[(((int)threadIdx.x) + 6016)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 94336)];
    kernel_shared[(((int)threadIdx.x) + 6144)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 98304)];
    kernel_shared[(((int)threadIdx.x) + 6272)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 98432)];
    kernel_shared[(((int)threadIdx.x) + 6400)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 102400)];
    kernel_shared[(((int)threadIdx.x) + 6528)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 102528)];
    kernel_shared[(((int)threadIdx.x) + 6656)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 106496)];
    kernel_shared[(((int)threadIdx.x) + 6784)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 106624)];
    kernel_shared[(((int)threadIdx.x) + 6912)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 110592)];
    kernel_shared[(((int)threadIdx.x) + 7040)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 110720)];
    kernel_shared[(((int)threadIdx.x) + 7168)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 114688)];
    kernel_shared[(((int)threadIdx.x) + 7296)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 114816)];
    kernel_shared[(((int)threadIdx.x) + 7424)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 118784)];
    kernel_shared[(((int)threadIdx.x) + 7552)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 118912)];
    kernel_shared[(((int)threadIdx.x) + 7680)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 122880)];
    kernel_shared[(((int)threadIdx.x) + 7808)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 123008)];
    kernel_shared[(((int)threadIdx.x) + 7936)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 126976)];
    kernel_shared[(((int)threadIdx.x) + 8064)] = kernel[((((k_outer_outer * 131072) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 127104)];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          T_matmul_NN_local[(i_c_outer_inner * 4)] = (T_matmul_NN_local[(i_c_outer_inner * 4)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31))]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 16)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 16)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 32)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 32)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 32)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 64)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 48)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 48)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 96)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 64)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 64)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 128)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 80)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 80)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 160)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 96)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 96)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 192)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 112)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 112)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 224)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 1)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[(((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31))]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 17)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 17)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 32)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 33)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 33)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 64)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 49)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 49)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 96)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 65)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 65)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 128)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 81)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 81)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 160)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 97)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 97)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 192)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 113)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 113)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 32)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 224)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 2)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 2)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[(((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31))]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 18)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 18)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 32)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 34)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 34)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 64)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 50)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 50)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 96)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 66)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 66)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 128)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 82)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 82)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 160)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 98)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 98)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 192)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 114)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 114)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 224)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 3)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 3)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[(((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31))]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 19)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 19)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 32)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 35)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 35)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 64)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 51)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 51)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 96)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 67)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 67)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 128)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 83)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 83)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 160)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 99)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 99)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 192)]));
          T_matmul_NN_local[((i_c_outer_inner * 4) + 115)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + 115)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 512) + (i_c_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 96)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 256)) + (((int)threadIdx.x) & 31)) + 224)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 32)] = T_matmul_NN_local[(i_inner + 16)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 64)] = T_matmul_NN_local[(i_inner + 32)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 96)] = T_matmul_NN_local[(i_inner + 48)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 128)] = T_matmul_NN_local[(i_inner + 64)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 160)] = T_matmul_NN_local[(i_inner + 80)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 192)] = T_matmul_NN_local[(i_inner + 96)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 31)) + 224)] = T_matmul_NN_local[(i_inner + 112)];
  }
}

