#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvmlPower.hpp"

#define N 128
#define CI 128
#define CM 1
#define H 28
#define W 28
#define KH 3
#define KW 3




int main(){
    int repeat = REPEAT;
    float warmupRatio = 0.1;
    float randMax = 1000.0;
    srand((unsigned)time(NULL));

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N * CI * H * W);
    b   = (float*)malloc(sizeof(float) * CI * CM * KH * KW);
    out = (float*)malloc(sizeof(float) * N * CI * H * W);

    // Initialize host arrays
    // memset(a,   1.0, sizeof(float) * N * CI * H * W);
    for(int i = 0; i < N * CI * H * W; i++) {
        a[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }
    // memset(b,   1.0, sizeof(float) * CI * CM * KH * KW);
    for(int i = 0; i < CI * CM * KH * KW; i++) {
        b[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a,   sizeof(float) * N * CI * H * W);
    hipMalloc((void**)&d_b,   sizeof(float) * CI * CM * KH * KW);
    hipMalloc((void**)&d_out, sizeof(float) * N * CI * H * W);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N * CI * H * W, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * CI * CM * KH * KW, hipMemcpyHostToDevice);

    // Executing kernel 
    for(int i = 0; i < repeat * warmupRatio; i++) {
        mydwconv_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    }
    hipDeviceSynchronize();
    nvmlAPIRun();
    for(int i = 0; i < repeat; i++) {
        mydwconv_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    }
    hipDeviceSynchronize();
    nvmlAPIEnd();
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N * CI * H * W, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
