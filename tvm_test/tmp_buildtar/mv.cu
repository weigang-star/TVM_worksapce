
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymv_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4];
  __shared__ float data_shared[8];
  __shared__ float kernel_shared[1024];
  for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
    T_matmul_NN_local[j_c_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_inner_init + 2)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 4) {
      *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + ((k_outer_outer * 8) + (((int)threadIdx.x) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + ((((k_outer_outer * 262144) + (ax0_ax1_fused_outer_outer * 32768)) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
        T_matmul_NN_local[j_c_inner] = (T_matmul_NN_local[j_c_inner] + (data_shared[k_inner] * kernel_shared[(((k_inner * 128) + (((int)threadIdx.x) * 2)) + j_c_inner)]));
        T_matmul_NN_local[(j_c_inner + 2)] = (T_matmul_NN_local[(j_c_inner + 2)] + (data_shared[k_inner] * kernel_shared[((((k_inner * 128) + (((int)threadIdx.x) * 2)) + j_c_inner) + 64)]));
      }
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    T_matmul_NN[(((((int)blockIdx.x) * 128) + (((int)threadIdx.x) * 2)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) * 2)) + j_inner) + 64)] = T_matmul_NN_local[(j_inner + 2)];
  }
}

