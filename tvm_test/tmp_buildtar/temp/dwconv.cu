
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(4) mydwconv_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[896];
  __shared__ float PaddedInput_shared[5760];
  __shared__ float kernel_shared[18];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 16; ++b_c_outer_inner_init) {
    for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
      for (int c_c_inner_init = 0; c_c_inner_init < 2; ++c_c_inner_init) {
        DepthwiseConv2d_local[(((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 128)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 256)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 384)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 512)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 640)] = 0.000000e+00f;
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 8) + (c_c_inner_init * 4)) + i_c_outer_inner_init) + 768)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 1440; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x))] = (((((1 <= (((((int)blockIdx.x) % 7) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 45) * 2) + (((int)threadIdx.x) >> 1)) / 15))) && ((((((int)blockIdx.x) % 7) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 45) * 2) + (((int)threadIdx.x) >> 1)) / 15)) < 29)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 30))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 30) < 29)) ? data[(((((((((((int)blockIdx.x) / 448) * 1605632) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 90) * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 90) / 45) * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 45) * 2) + (((int)threadIdx.x) >> 1)) / 15) * 28)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 30)) - 29)] : 0.000000e+00f);
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 2) + (((int)threadIdx.x) >> 1)) < 9) {
      kernel_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = kernel[(((((((int)blockIdx.x) % 448) / 7) * 18) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_1 * 4)) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int b_c_outer_inner = 0; b_c_outer_inner < 16; ++b_c_outer_inner) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int di_inner = 0; di_inner < 3; ++di_inner) {
        for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
          for (int c_c_inner = 0; c_c_inner < 2; ++c_c_inner) {
            DepthwiseConv2d_local[(((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner)] = (DepthwiseConv2d_local[(((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner)] + (PaddedInput_shared[((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 128)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 128)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 4)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 256)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 256)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 8)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 384)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 384)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 12)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 512)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 512)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 16)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 640)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 640)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 20)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
            DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 768)] = (DepthwiseConv2d_local[((((b_c_outer_inner * 8) + (c_c_inner * 4)) + i_c_outer_inner) + 768)] + (PaddedInput_shared[(((((((b_c_outer_inner * 360) + (c_c_inner * 180)) + (i_c_outer_inner * 30)) + (di_inner * 30)) + ((int)threadIdx.x)) + dj_inner) + 24)] * kernel_shared[(((c_c_inner * 9) + (di_inner * 3)) + dj_inner)]));
          }
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 16; ++b_inner) {
    for (int c_inner = 0; c_inner < 2; ++c_inner) {
      for (int i_inner = 0; i_inner < 4; ++i_inner) {
        DepthwiseConv2d[((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x))] = DepthwiseConv2d_local[(((b_inner * 8) + (c_inner * 4)) + i_inner)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 4)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 128)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 8)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 256)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 12)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 384)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 16)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 512)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 20)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 640)];
        DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 1605632) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 7) * 1568)) + (c_inner * 784)) + ((((int)blockIdx.x) % 7) * 112)) + (i_inner * 28)) + ((int)threadIdx.x)) + 24)] = DepthwiseConv2d_local[((((b_inner * 8) + (c_inner * 4)) + i_inner) + 768)];
      }
    }
  }
}

