
#include <hip/hip_runtime.h>
// 16384 1 1
// 128 1 1

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  // block:

  // load [256*1024] from data (256 rows) (65536/256=256)
  // load [1024*64] from kernel (64 cols) (4096/64=64)
  // compute [256*1024] x [1024*64] = [256*64]

  // K for 128: 
  // load [256*8] (data_shared[2048]) (* 128)
  // load [8*64] (kernel_shared[512]) (128 *)
  // compute [256*8] x [8*64] = [256*64] (128 threads * T_matmul_NN_local[128])

  // write back [256*64] to T_matmul_NN   (65536/256) * (4096/64) = 256*64 = 16384 = gridDim

  float T_matmul_NN_local[128];         // result
  __shared__ float data_shared[2048];   // data shared memory
  __shared__ float kernel_shared[512];  // kernel shared memory

  // Init
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 8; ++j_c_inner_init) {
      T_matmul_NN_local[(((i_c_inner_init * 8) + j_c_inner_init) + 0 )] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 8) + j_c_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 8) + j_c_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 8) + j_c_inner_init) + 96)] = 0.000000e+00f;
    }
  }

  // [4, 1024] x [1024, 8]
  // K 1024 / 128 = 8

  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();

    // data -> data shared
    // 1 thread: 2 float 
    // 128 threads: tid[:2]*1024(K), tid[1:0]*2, 128(32*4) threads -> [32*8] from data
    // 8 line: 32768 = 32*1024 -> 8 * [32*8] -> [256*8] from data (data_shared[2048])
    // block-for: k_outer_outer[0-128)*8 -> [256*8] * 128 -> [256*1024] from data -> 256 rows from data
    // M: line_count * 2^(tid_high) = 8 * 2^4 = 8*16 = 128
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 0   )) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 0     ));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 256 )) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 32768 ));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 512 )) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 65536 ));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 768 )) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 98304 ));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 131072));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 163840));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 196608));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 229376));
    
    // kernel -> kernel shared  4*128 = 512
    // 1 thread: 4 float
    // 128 threads: tid[:4]*4096(N), tid[3:0]*4, 128(8*16) threads -> [8*64] from kernel (kernel_shared[512])
    // block-for: k_outer_outer[0-128)*32768(8*4096(N)) -> 128 * [8*64] -> [1024*64] from kernel -> 64 cols from kernel
    // N: g * line_count * 2^(tid_low) = 1 * 1 * 2^4 = 16
    *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + ((((k_outer_outer * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    __syncthreads();
    
    // compute
    // 1 line: [4*8]
    // 4 line: 4 * [4*8]
    // 128 threads: 128 * 4 * [4*8]

    // 
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 8; ++j_c_inner) {
          T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 0 )] += (/*T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 0 )]*/ + (data_shared[(((((((int)threadIdx.x) >> 3) * 32) + (i_c_inner * 8)) + k_inner) + 0   )]   * kernel_shared[(((k_inner * 64) + ((((int)threadIdx.x) & 7) * 8)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 32)] += (/*T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 32)]*/ + (data_shared[(((((((int)threadIdx.x) >> 3) * 32) + (i_c_inner * 8)) + k_inner) + 512 )]   * kernel_shared[(((k_inner * 64) + ((((int)threadIdx.x) & 7) * 8)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 64)] += (/*T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 64)]*/ + (data_shared[(((((((int)threadIdx.x) >> 3) * 32) + (i_c_inner * 8)) + k_inner) + 1024)]   * kernel_shared[(((k_inner * 64) + ((((int)threadIdx.x) & 7) * 8)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 96)] += (/*T_matmul_NN_local[(((i_c_inner * 8) + j_c_inner) + 96)]*/ + (data_shared[(((((((int)threadIdx.x) >> 3) * 32) + (i_c_inner * 8)) + k_inner) + 1536)]   * kernel_shared[(((k_inner * 64) + ((((int)threadIdx.x) & 7) * 8)) + j_c_inner)]));
        }
      }
    }
  }

  // Write back
  // j_inner [0,8) -> [1*8] 
  // i_inner [0,4) -> [4*8]
  // 1 thread: [4*8] float
  // 128 threads: tid[:3]*16384(4*4096), tid[2:0]*8, 128(16*8) threads -> [16*8] x [4*8] -> [64*64]
  // 4 line: 262144=64*4096 -> 4 * [64*64] -> [256*64] to T_matmul_NN
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + j_inner) + 0     )] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 0 )];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + j_inner) + 786432)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 96)];
    }
  }
}

