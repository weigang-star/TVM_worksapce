
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 16; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 32)) + j_c_inner_init)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 128; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 8192) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 32; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer_1 >> 2) * 4096)) + (((int)blockIdx.x) * 256)) + ((ax0_ax1_fused_outer_outer_1 & 3) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 16; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 2; ++k_inner) {
            for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
              for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
                T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 32)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 32)) + j_c_inner)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 32)) + (i_c_inner * 8)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((((k_outer_inner * 512) + (k_inner * 256)) + ((((int)threadIdx.x) & 3) * 64)) + (j_c_outer_inner * 32)) + j_c_inner)]));
              }
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 64; ++i_inner) {
    for (int j_inner = 0; j_inner < 64; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 2) * 262144) + (i_inner * 4096)) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.x) & 3) * 64)) + j_inner)] = T_matmul_NN_local[((i_inner * 64) + j_inner)];
    }
  }
}

