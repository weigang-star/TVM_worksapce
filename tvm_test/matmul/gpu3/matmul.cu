
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 64; ++i_c_outer_inner_init) {
    T_matmul_NN_local[(i_c_outer_inner_init * 16)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 8)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 1)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 9)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 2)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 10)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 3)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 11)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 4)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 12)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 5)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 13)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 6)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 14)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 7)] = 0.000000e+00f;
    T_matmul_NN_local[((i_c_outer_inner_init * 16) + 15)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + (((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 65536));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 131072));
    *(float2*)(data_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(data + ((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 196608));
    kernel_shared[((int)threadIdx.x)] = kernel[(((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 128)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 256)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 384)];
    kernel_shared[(((int)threadIdx.x) + 512)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 640)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 4224)];
    kernel_shared[(((int)threadIdx.x) + 768)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 4352)];
    kernel_shared[(((int)threadIdx.x) + 896)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 4480)];
    kernel_shared[(((int)threadIdx.x) + 1024)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 1152)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 8320)];
    kernel_shared[(((int)threadIdx.x) + 1280)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 8448)];
    kernel_shared[(((int)threadIdx.x) + 1408)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 8576)];
    kernel_shared[(((int)threadIdx.x) + 1536)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 1664)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 12416)];
    kernel_shared[(((int)threadIdx.x) + 1792)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 12544)];
    kernel_shared[(((int)threadIdx.x) + 1920)] = kernel[((((k_outer_outer * 16384) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x)) + 12672)];
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 64; ++i_c_outer_inner) {
      T_matmul_NN_local[(i_c_outer_inner * 16)] = (T_matmul_NN_local[(i_c_outer_inner * 16)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[((((int)threadIdx.x) & 63) * 8)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[((((int)threadIdx.x) & 63) * 8)]));
      T_matmul_NN_local[(i_c_outer_inner * 16)] = (T_matmul_NN_local[(i_c_outer_inner * 16)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 512)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 512)]));
      T_matmul_NN_local[(i_c_outer_inner * 16)] = (T_matmul_NN_local[(i_c_outer_inner * 16)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1024)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1024)]));
      T_matmul_NN_local[(i_c_outer_inner * 16)] = (T_matmul_NN_local[(i_c_outer_inner * 16)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1536)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 8)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1536)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 513)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 513)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1025)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1025)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 1)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1537)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 9)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1537)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 2)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 2)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 514)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 514)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1026)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1026)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 2)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1538)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 10)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1538)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 3)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 3)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 515)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 515)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1027)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1027)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 3)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1539)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 11)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1539)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 4)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 4)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 516)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 516)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1028)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1028)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 4)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1540)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 12)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1540)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 5)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 5)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 517)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 517)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1029)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1029)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 5)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1541)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 13)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1541)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 6)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 6)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 518)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 518)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1030)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1030)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 6)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1542)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 14)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1542)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] + (data_shared[(((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8))] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 7)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 4)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 7)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 1)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 519)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 5)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 519)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 2)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1031)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 6)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1031)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 7)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 3)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1543)]));
      T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + 15)] + (data_shared[((((((int)threadIdx.x) >> 6) * 512) + (i_c_outer_inner * 8)) + 7)] * kernel_shared[(((((int)threadIdx.x) & 63) * 8) + 1543)]));
    }
  }
  for (int i_inner = 0; i_inner < 128; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 6) * 524288)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
    }
  }
}

