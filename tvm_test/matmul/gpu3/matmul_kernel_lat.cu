#define REPEAT_WARMUP 87
#define REPEAT_RUN 1
dim3 dimGrid(32, 1, 1);
dim3 dimBlock(128, 1, 1);
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN);

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define M 1024
#define N 4096
#define K 1024

__global__ void warm_up_gpu(long seed, float* __restrict__ temp){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    float it = 0.0f;
    for(int i = 0; i < 5000000; i++)
        it += idx * (hiprand_uniform_double(&state) - 0.5);
    temp[0] = it;
}

int main() {
    hipEvent_t warmup, start, stop;
    hipEventCreate(&warmup);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float randMax = 1000.0;
    srand((unsigned)time(NULL));
    
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 
    // float *d_temp;

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * M * K);
    b   = (float*)malloc(sizeof(float) * N * K);
    out = (float*)malloc(sizeof(float) * M * N);

    // Initialize host arrays
    for(int i = 0; i < M * K; i++) {
        a[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }
    for(int i = 0; i < N * K; i++) {
        b[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a,   sizeof(float) * M * K);
    hipMalloc((void**)&d_b,   sizeof(float) * N * K);
    hipMalloc((void**)&d_out, sizeof(float) * M * N);
    // cudaMalloc((void**)&d_temp, sizeof(float) * 1);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * K, hipMemcpyHostToDevice);

    // Executing kernel 
    hipEventRecord(warmup);
    // warm_up_gpu<<<dim3(256,1,1), dim3(256,1,1)>>>(rand(), d_temp);
    for(int i = 0; i < REPEAT_WARMUP; i++) mymatmul_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    hipEventRecord(start);
    for(int i = 0; i < REPEAT_RUN; i++) mymatmul_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Transfer data back to host memory
    // cudaMemcpy(out, d_out, sizeof(float) * N * M, cudaMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

    float timeWarmup = 0;
    float timeRun = 0;
    hipEventElapsedTime(&timeWarmup, warmup, start);
    hipEventElapsedTime(&timeRun, start, stop);
    // printf("warmup time= %lf ms, run time = %lf ms\n", timeWarmup, timeRun);
    printf("%f\n", timeRun/REPEAT_RUN);
}
