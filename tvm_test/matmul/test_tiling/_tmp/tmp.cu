
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[256];
  __shared__ float data_shared[256];
  __shared__ float kernel_shared[8192];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 2) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 96)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 128)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 160)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 192)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 224)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      *(float4*)(data_shared + (((int)threadIdx.x) * 4)) = *(float4*)(data + (((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)));
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer >> 3) * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((ax0_ax1_fused_outer_outer & 7) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
          T_matmul_NN_local[((i_c_inner * 2) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 2) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 32)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 32)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 64)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 256)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 96)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 96)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 384)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 128)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 128)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 512)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 160)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 160)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 640)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 192)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 192)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 768)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 224)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 224)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 896)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 96)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 640)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 160)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 768)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 192)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 896)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 224)];
    }
  }
}

