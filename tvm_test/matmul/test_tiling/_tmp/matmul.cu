
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[1024];
  for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 2) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 4)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 8)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 12)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 16)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 20)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 24)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 28)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 7) * 65536) + (ax0_ax1_fused_outer_outer * 2048)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 131072) + (ax0_ax1_fused_outer_outer_1 * 8192)) + ((((int)threadIdx.x) >> 5) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 31))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 8; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[((i_c_inner * 2) + j_c_inner)] = (T_matmul_NN_local[((i_c_inner * 2) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 4)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 4)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 8)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 8)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 512)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 12)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 12)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 512)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 16)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 16)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 1024)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 20)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 20)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 1024)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 24)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 24)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 1536)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 28)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 28)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 32)) + (k_outer_inner * 8)) + k_inner) + 1536)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 16)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 4)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 65536)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 8)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 65552)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 12)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 131088)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 20)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 196608)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 24)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 196624)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 28)];
    }
  }
}

