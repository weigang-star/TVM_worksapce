
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 2) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 8)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 16)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_outer_inner_init) + 24)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    // k = 1024 / 512 = 2

    // M_size = 32 / 2 = 16
    // M = 512 / 16 = 32
    // [16*2]
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 3) * 16384) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }

    // N_size = 1024 / 2 = 512
    // N = 4096 / 512 = 8
    // [2*512]
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 512) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + ((((k_outer_outer * 8192) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();

    // block [16*512]
    // thread [4*8]
    // M = 16 / 4 = 4
    // N = 512 / 8 = 64
    for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 0 )] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 0 )] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 0  )]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 8 )] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 8 )] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 16)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 16)] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 256)]));
          T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 24)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_outer_inner) + 24)] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 384)]));
        }
      }
    }
  }

  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 8)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 24)];
    }
  }
}
