
#include <hip/hip_runtime.h>
// 1024 1 1
// 256 1 1

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  // block:

  // load [128*1024] from data (128 rows) (512/128=4)
  // load [1024*16] from kernel (16 cols) (4096/16=256)
  // compute [128*1024] x [1024*16] = [128*16]

  // K for 64: 
  // load [128*16] (data_shared[2048]) (* 64)
  // load [16*16] (kernel_shared[512]) (64 *)
  // compute [128*16] x [16*16] = [128*16] (256 threads * T_matmul_NN_local[8])
  float T_matmul_NN_local[8];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[256];

  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    T_matmul_NN_local[j_c_outer_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 4)] = 0.000000e+00f;
  }

  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    // 1 thread: 1 float 
    // 128 threads: tid[:4]*1024(K), tid[3:0]*1, 256(16*16) threads -> [16*16] from data
    // 8 line: 16384 = 16*1024 -> 8 * [16*16] -> [128*16] from data (data_shared[2048])
    // block-for: k_outer_outer[0-64)*16 -> [128*16] * 64 -> [128*1024] from data -> 128 rows from data
    // M_size: line_count * 2^(tid_high) = 8 * 2^4 = 8*16 = 128
    // M = 512 / 128 = 4
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 8) * 131072) + (ax0_ax1_fused_outer_outer * 16384)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }

    // kernel -> kernel shared
    // 1 thread: 1 float
    // 128 threads: tid[:4]*4096(N), tid[3:0]*1, 256(16*16) threads -> [16*16] from kernel (kernel_shared[256])
    // block-for: k_outer_outer[0-64)*65536(16*4096(N)) -> 64 * [16*16] -> [1024*16] from kernel -> 16 cols from kernel
    // N_size: g * line_count * 2^(tid_low) = 1 * 1 * 2^4 = 16
    // N = 4096 / 16 = 256
    kernel_shared[((int)threadIdx.x)] = kernel[((((k_outer_outer * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 15))];
    __syncthreads();

    // M: line_count * 2^(tid_high) = 1 * 2^6 = 64
    // N: g * line_count * 2^(tid_low) = 1 * 1 * 2^2 = 4
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 4; ++k_inner) {
          T_matmul_NN_local[(j_c_outer_inner + 0)] = (T_matmul_NN_local[(j_c_outer_inner + 0)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 4)) + k_inner) + 0   )] * kernel_shared[((((k_outer_inner * 64) + (k_inner * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
          T_matmul_NN_local[(j_c_outer_inner + 4)] = (T_matmul_NN_local[(j_c_outer_inner + 4)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 4)) + k_inner) + 1024)] * kernel_shared[((((k_outer_inner * 64) + (k_inner * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
        }
      }
    }
  }

  for (int j_inner = 0; j_inner < 4; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 8) * 524288) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 8) * 524288) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 262144)] = T_matmul_NN_local[(j_inner + 4)];
  }
}

