
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[8192];
  for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 32) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 32) + j_c_inner_init) + 64)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 1024) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 32768) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
            T_matmul_NN_local[((i_c_inner * 32) + j_c_inner)] = (T_matmul_NN_local[((i_c_inner * 32) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 16) + (i_c_inner * 8)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 1024)) + ((((int)threadIdx.x) & 31) * 32)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 32) + j_c_inner) + 64)] = (T_matmul_NN_local[(((i_c_inner * 32) + j_c_inner) + 64)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_c_inner * 8)) + (k_outer_inner * 2)) + k_inner) + 512)] * kernel_shared[((((k_outer_inner * 2048) + (k_inner * 1024)) + ((((int)threadIdx.x) & 31) * 32)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 32; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 524288) + ((((int)threadIdx.x) >> 5) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 32)) + j_inner)] = T_matmul_NN_local[((i_inner * 32) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 524288) + ((((int)threadIdx.x) >> 5) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 32)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 64)];
    }
  }
}

