
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 2) + j_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 5) * 2048) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + ((((k_outer_outer * 65536) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 4; ++k_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] + (data_shared[(((i_c_outer_inner * 16) + (k_outer_inner * 4)) + k_inner)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 5) * 8192) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + (((int)threadIdx.x) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
    }
  }
}

