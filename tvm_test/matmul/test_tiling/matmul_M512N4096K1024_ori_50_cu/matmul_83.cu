
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[4096];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 128; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 128) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 128) + j_c_outer_inner_init) + 2048)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 8192) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 32; ++ax0_ax1_fused_outer_outer_1) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 128) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + (((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer_1 >> 2) * 4096)) + (((int)blockIdx.x) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 3) * 128)) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 128; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
          T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 1) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 512) + ((((int)threadIdx.x) & 1) * 128)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 2048)] = (T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 1) * 128) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 512) + ((((int)threadIdx.x) & 1) * 128)) + j_c_outer_inner) + 256)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 1) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 512)) + ((((int)threadIdx.x) & 1) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 1) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 512)) + ((((int)threadIdx.x) & 1) * 128)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 2048)];
    }
  }
}

