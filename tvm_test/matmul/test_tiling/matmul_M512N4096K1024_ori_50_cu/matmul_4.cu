
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[4096];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 1024)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 3072)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 4096)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 5120)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 6144)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 7168)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 8; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 64) {
        data_shared[((((int)threadIdx.x) * 8) + ax0_ax1_fused_inner_s)] = data[(((((int)threadIdx.x) * 8192) + (ax0_ax1_fused_inner_s * 1024)) + k_outer_outer)];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((k_outer_outer * 4096) + (ax0_ax1_fused_outer_outer * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
          T_matmul_NN_local[(((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner)] + (data_shared[(((((int)threadIdx.x) >> 4) * 8) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 1024)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 1024)] + (data_shared[(((((int)threadIdx.x) >> 4) * 8) + i_c_inner)] * kernel_shared[(((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 128)] * kernel_shared[((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 3072)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 3072)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 128)] * kernel_shared[(((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 4096)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 4096)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 256)] * kernel_shared[((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 5120)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 5120)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 256)] * kernel_shared[(((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 6144)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 6144)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 384)] * kernel_shared[((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 7168)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 7168)] + (data_shared[((((((int)threadIdx.x) >> 4) * 8) + i_c_inner) + 384)] * kernel_shared[(((((((int)threadIdx.x) & 15) * 128) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[(((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 1024)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 2048)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 526336)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 3072)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 4096)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 1050624)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 5120)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 1572864)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 6144)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 32768) + (i_inner * 4096)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 1574912)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 7168)];
    }
  }
}

