
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[64];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[4096];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 2) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 8)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 16)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 24)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 40)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 48)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 2) + j_c_inner_init) + 56)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(data_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 4))) = *(float4*)(data + (((((((int)blockIdx.x) >> 7) * 65536) + (ax0_ax1_fused_outer_outer * 1024)) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 64) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + (((((k_outer_outer * 524288) + (ax0_ax1_fused_outer_outer_1 * 8192)) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 15) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 8; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[((i_c_inner * 2) + j_c_inner)] = (T_matmul_NN_local[((i_c_inner * 2) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 8)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 8)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 16)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 16)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 2048)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 24)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 24)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 2048)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 32)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 32)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 4096)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 40)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 40)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 4096)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 48)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 48)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 6144)] * kernel_shared[((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 56)] = (T_matmul_NN_local[(((i_c_inner * 2) + j_c_inner) + 56)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 6144)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 16)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 8)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 65536)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 65552)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 24)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 131088)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 40)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 196608)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 48)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 196624)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 56)];
    }
  }
}

