
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[2048];
  __shared__ float data_shared[256];
  __shared__ float kernel_shared[512];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 128; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 8) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 1024)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 16; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 16) {
        data_shared[((((int)threadIdx.x) * 16) + ax0_ax1_fused_inner_s)] = data[(((((((int)blockIdx.x) >> 3) * 262144) + (((int)threadIdx.x) * 16384)) + (ax0_ax1_fused_inner_s * 1024)) + k_outer_outer)];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 512)) + (ax0_ax1_fused_outer_outer * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 128; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
        T_matmul_NN_local[((i_c_outer_inner * 8) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 8) + j_c_outer_inner)] + (data_shared[(((((int)threadIdx.x) >> 5) * 128) + i_c_outer_inner)] * kernel_shared[(((((int)threadIdx.x) & 31) * 8) + j_c_outer_inner)]));
        T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 1024)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 1024)] + (data_shared[(((((int)threadIdx.x) >> 5) * 128) + i_c_outer_inner)] * kernel_shared[((((((int)threadIdx.x) & 31) * 8) + j_c_outer_inner) + 256)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 128; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 5) * 524288)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 31) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 5) * 524288)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 31) * 8)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 1024)];
    }
  }
}

