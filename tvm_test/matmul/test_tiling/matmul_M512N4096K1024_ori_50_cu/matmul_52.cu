
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[256];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[512];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 8) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 8) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 128)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(data_shared + ((ax0_ax1_fused_outer_outer * 256) + (((int)threadIdx.x) * 2))) = *(float2*)(data + ((((((((int)blockIdx.x) >> 5) * 262144) + (ax0_ax1_fused_outer_outer * 65536)) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 4; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 16384) + (ax0_ax1_fused_outer_outer_1 * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 4) * 64) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner) + 128)] = (T_matmul_NN_local[((((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner) + 128)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 64) + (i_c_inner * 4)) + k_outer_inner) + 512)] * kernel_shared[((((k_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 5) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 128)];
    }
  }
}

