
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[16];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[512];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    T_matmul_NN_local[i_c_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 4)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 8)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 12)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 8; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 4) {
        data_shared[((((int)threadIdx.x) * 8) + ax0_ax1_fused_inner_s)] = data[((((((((int)blockIdx.x) >> 4) * 16384) + (((int)threadIdx.x) * 4096)) + ((ax0_ax1_fused_inner_s >> 1) * 1024)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s & 1))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 8192) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        T_matmul_NN_local[i_c_inner] = (T_matmul_NN_local[i_c_inner] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((k_inner * 256) + (((int)threadIdx.x) & 63))]));
        T_matmul_NN_local[(i_c_inner + 4)] = (T_matmul_NN_local[(i_c_inner + 4)] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 256) + (((int)threadIdx.x) & 63)) + 64)]));
        T_matmul_NN_local[(i_c_inner + 8)] = (T_matmul_NN_local[(i_c_inner + 8)] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 256) + (((int)threadIdx.x) & 63)) + 128)]));
        T_matmul_NN_local[(i_c_inner + 12)] = (T_matmul_NN_local[(i_c_inner + 12)] + (data_shared[((((((int)threadIdx.x) >> 6) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 256) + (((int)threadIdx.x) & 63)) + 192)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 63))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 63)) + 64)] = T_matmul_NN_local[(i_inner + 4)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 63)) + 128)] = T_matmul_NN_local[(i_inner + 8)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) & 63)) + 192)] = T_matmul_NN_local[(i_inner + 12)];
  }
}

