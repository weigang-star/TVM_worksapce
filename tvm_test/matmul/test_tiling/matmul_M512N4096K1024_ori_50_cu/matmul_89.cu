
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[16];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    T_matmul_NN_local[j_c_outer_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 4)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 8)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 12)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 6) * 32768) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 8; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + (ax0_ax1_fused_outer_outer_1 * 8192)) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + (((int)threadIdx.x) & 63))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          T_matmul_NN_local[j_c_outer_inner] = (T_matmul_NN_local[j_c_outer_inner] + (data_shared[((((((int)threadIdx.x) >> 3) * 16) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
          T_matmul_NN_local[(j_c_outer_inner + 4)] = (T_matmul_NN_local[(j_c_outer_inner + 4)] + (data_shared[((((((int)threadIdx.x) >> 3) * 16) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
          T_matmul_NN_local[(j_c_outer_inner + 8)] = (T_matmul_NN_local[(j_c_outer_inner + 8)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 16) + (k_outer_inner * 2)) + k_inner) + 256)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
          T_matmul_NN_local[(j_c_outer_inner + 12)] = (T_matmul_NN_local[(j_c_outer_inner + 12)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 16) + (k_outer_inner * 2)) + k_inner) + 256)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 4; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 32)] = T_matmul_NN_local[(j_inner + 4)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 65536)] = T_matmul_NN_local[(j_inner + 8)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 65568)] = T_matmul_NN_local[(j_inner + 12)];
  }
}

