
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_outer_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 64)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[(((((int)blockIdx.x) * 32768) + (((int)threadIdx.x) * 1024)) + k_outer_outer)];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 1024) + ((int)threadIdx.x))] = kernel[(((k_outer_outer * 4096) + (ax0_ax1_fused_outer_outer * 1024)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_outer_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 9) * 16) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[(((((int)threadIdx.x) & 511) * 4) + j_c_outer_inner)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_outer_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_outer_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 9) * 16) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 511) * 4) + j_c_outer_inner) + 2048)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((int)blockIdx.x) * 131072) + ((((int)threadIdx.x) >> 9) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 511) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((int)blockIdx.x) * 131072) + ((((int)threadIdx.x) >> 9) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 511) * 4)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 64)];
    }
  }
}

