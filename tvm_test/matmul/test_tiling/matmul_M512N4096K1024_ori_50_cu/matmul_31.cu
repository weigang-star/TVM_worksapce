
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[128];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (i_c_inner_init * 32)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 32)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 1024)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 32)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 32)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 3072)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 16) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 1) * 32768) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 512; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 16) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 7) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((ax0_ax1_fused_outer_outer_1 & 127) * 16)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 2; ++k_inner) {
            for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
              for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
                T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((((k_outer_inner * 4096) + (k_inner * 2048)) + (((int)threadIdx.x) * 32)) + (j_c_outer_inner * 4)) + j_c_inner)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)] + (data_shared[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 2048)) + (((int)threadIdx.x) * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 512)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] + (data_shared[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 2048)) + (((int)threadIdx.x) * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 3072)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 3072)] + (data_shared[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 2048)) + (((int)threadIdx.x) * 32)) + (j_c_outer_inner * 4)) + j_c_inner) + 1536)]));
              }
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 32; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 1) * 131072) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + (((int)threadIdx.x) * 32)) + j_inner)] = T_matmul_NN_local[((i_inner * 32) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 131072) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + (((int)threadIdx.x) * 32)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 1024)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 131072) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + (((int)threadIdx.x) * 32)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 2048)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 131072) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + (((int)threadIdx.x) * 32)) + j_inner) + 1536)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 3072)];
    }
  }
}

