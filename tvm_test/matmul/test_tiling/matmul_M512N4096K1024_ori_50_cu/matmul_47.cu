
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[64];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 4) + i_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 4) + i_c_inner_init) + 16)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 4) * 16384) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((ax0_ax1_fused_outer_outer & 1) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 4) + i_c_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 4) + i_c_inner)] + (data_shared[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((k_outer_inner * 512) + (k_inner * 256)) + ((int)threadIdx.x))]));
            T_matmul_NN_local[(((i_c_outer_inner * 4) + i_c_inner) + 16)] = (T_matmul_NN_local[(((i_c_outer_inner * 4) + i_c_inner) + 16)] + (data_shared[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 256)) + ((int)threadIdx.x)) + 128)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    T_matmul_NN[(((((((int)blockIdx.x) >> 4) * 65536) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 65536) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((int)threadIdx.x)) + 128)] = T_matmul_NN_local[(i_inner + 16)];
  }
}

