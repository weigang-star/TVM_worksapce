
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init) + 32)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init) + 64)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init) + 96)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 6) * 262144) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 131072) + (ax0_ax1_fused_outer_outer_1 * 8192)) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + (((int)threadIdx.x) & 63))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int k_inner = 0; k_inner < 32; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_outer_inner * 128)) + (i_c_inner * 32)) + k_inner)] * kernel_shared[(((k_inner * 64) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 32)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 32)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_outer_inner * 128)) + (i_c_inner * 32)) + k_inner)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 16)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 64)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_outer_inner * 128)) + (i_c_inner * 32)) + k_inner)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 32)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 96)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 96)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_outer_inner * 128)) + (i_c_inner * 32)) + k_inner)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 7) * 2)) + j_c_inner) + 48)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 16)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 32)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 3) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 48)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 96)];
    }
  }
}

