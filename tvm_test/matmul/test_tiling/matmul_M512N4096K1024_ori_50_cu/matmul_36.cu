
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[256];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[512];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_outer_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 64)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 128)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_outer_inner_init) + 192)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 64; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 32) {
        data_shared[((((int)threadIdx.x) * 64) + ax0_ax1_fused_inner_s)] = data[((((((((int)blockIdx.x) >> 6) * 262144) + (((int)threadIdx.x) * 8192)) + ((ax0_ax1_fused_inner_s >> 3) * 1024)) + (k_outer_outer * 8)) + (ax0_ax1_fused_inner_s & 7))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + (((((k_outer_outer * 32768) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 5) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 31) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 64)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 16)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 128)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 128)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 32)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 192)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_outer_inner) + 192)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 48)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 16)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 32)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 48)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 192)];
    }
  }
}

