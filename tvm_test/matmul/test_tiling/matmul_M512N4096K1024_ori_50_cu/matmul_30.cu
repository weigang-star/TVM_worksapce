
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[2048];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[8192];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 128; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 128) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 128) + j_c_outer_inner_init) + 512)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 128) + j_c_outer_inner_init) + 1024)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 128) + j_c_outer_inner_init) + 1536)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 4; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 128) {
        data_shared[((((int)threadIdx.x) * 4) + ax0_ax1_fused_inner_s)] = data[(((((((int)blockIdx.x) * 262144) + (((int)threadIdx.x) * 2048)) + ((ax0_ax1_fused_inner_s >> 1) * 1024)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s & 1))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + (((k_outer_outer * 8192) + (ax0_ax1_fused_outer_outer * 2048)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 128; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 3) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 4096) + ((((int)threadIdx.x) & 7) * 128)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 512)] = (T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 512)] + (data_shared[((((((int)threadIdx.x) >> 3) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 4096) + ((((int)threadIdx.x) & 7) * 128)) + j_c_outer_inner) + 1024)]));
          T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 1024)] = (T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 1024)] + (data_shared[((((((int)threadIdx.x) >> 3) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 4096) + ((((int)threadIdx.x) & 7) * 128)) + j_c_outer_inner) + 2048)]));
          T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 1536)] = (T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 1536)] + (data_shared[((((((int)threadIdx.x) >> 3) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 4096) + ((((int)threadIdx.x) & 7) * 128)) + j_c_outer_inner) + 3072)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[(((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 512)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 1024)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner) + 3072)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 1536)];
    }
  }
}

