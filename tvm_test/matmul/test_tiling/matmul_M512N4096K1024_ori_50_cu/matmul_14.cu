
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[128];
  __shared__ float kernel_shared[512];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 16; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 16) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 96)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((ax0_ax1_fused_outer_outer & 1) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_inner = 0; j_c_inner < 16; ++j_c_inner) {
          T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 2) * 4) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 256) + ((((int)threadIdx.x) & 3) * 16)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 32)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 32)] + (data_shared[((((((int)threadIdx.x) >> 2) * 4) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 3) * 16)) + j_c_inner) + 64)]));
          T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 64)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 2) * 4) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 3) * 16)) + j_c_inner) + 128)]));
          T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 96)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 96)] + (data_shared[((((((int)threadIdx.x) >> 2) * 4) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 3) * 16)) + j_c_inner) + 192)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 3) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 3) * 16)) + j_inner) + 64)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 3) * 16)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 3) * 16)) + j_inner) + 192)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 96)];
    }
  }
}

