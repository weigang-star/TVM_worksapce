
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 16)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 32)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 48)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 64)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 80)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 96)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 112)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 64; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 8) {
        data_shared[((((int)threadIdx.x) * 64) + ax0_ax1_fused_inner_s)] = data[((((((((int)blockIdx.x) >> 3) * 65536) + (((int)threadIdx.x) * 8192)) + ((ax0_ax1_fused_inner_s >> 3) * 1024)) + (k_outer_outer * 8)) + (ax0_ax1_fused_inner_s & 7))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer & 1) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner)] * kernel_shared[(((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 16)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 16)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 32)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 32)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 256)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 48)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 48)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 384)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 64)] + (data_shared[((((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner) + 256)] * kernel_shared[(((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 80)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 80)] + (data_shared[((((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner) + 256)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 96)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 96)] + (data_shared[((((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner) + 256)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 256)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 112)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 112)] + (data_shared[((((((((int)threadIdx.x) >> 6) * 64) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_inner) + 256)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 384)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 48)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131200)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 80)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131328)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 96)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 6) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 131456)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 112)];
    }
  }
}

