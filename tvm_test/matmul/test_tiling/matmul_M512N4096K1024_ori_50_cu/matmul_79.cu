
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[8192];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 64; ++j_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
      T_matmul_NN_local[((j_c_outer_inner_init * 2) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 128)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 256)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 384)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 512)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 640)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 768)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 2) + j_c_inner_init) + 896)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + (((((((int)blockIdx.x) >> 1) * 131072) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)));
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 1024) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + (((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((ax0_ax1_fused_outer_outer & 1) * 1024)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 64; ++j_c_outer_inner) {
        for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
          T_matmul_NN_local[((j_c_outer_inner * 2) + j_c_inner)] = (T_matmul_NN_local[((j_c_outer_inner * 2) + j_c_inner)] + (data_shared[(((((int)threadIdx.x) >> 4) * 4) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 128)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 128)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 64)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 256)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 256)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 128)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 384)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 384)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 192)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 512)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 512)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 256)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 640)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 640)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 320)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 768)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 768)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 384)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 896)] = (T_matmul_NN_local[(((j_c_outer_inner * 2) + j_c_inner) + 896)] + (data_shared[((((((int)threadIdx.x) >> 4) * 4) + k_outer_inner) + 448)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 15) * 128)) + (j_c_outer_inner * 2)) + j_c_inner)]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 128; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 65536)] = T_matmul_NN_local[(j_inner + 128)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 131072)] = T_matmul_NN_local[(j_inner + 256)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 196608)] = T_matmul_NN_local[(j_inner + 384)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 262144)] = T_matmul_NN_local[(j_inner + 512)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 327680)] = T_matmul_NN_local[(j_inner + 640)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 393216)] = T_matmul_NN_local[(j_inner + 768)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + j_inner) + 458752)] = T_matmul_NN_local[(j_inner + 896)];
  }
}

