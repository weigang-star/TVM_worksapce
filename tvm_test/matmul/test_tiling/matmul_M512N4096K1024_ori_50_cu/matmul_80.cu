
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[16];
  __shared__ float data_shared[16];
  __shared__ float kernel_shared[512];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
    T_matmul_NN_local[j_c_outer_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 2)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 4)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 6)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 8)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 10)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 12)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 14)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 8) {
      *(float2*)(data_shared + (((int)threadIdx.x) * 2)) = *(float2*)(data + ((((((int)blockIdx.x) >> 4) * 8192) + (((int)threadIdx.x) * 1024)) + (k_outer_outer * 2)));
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((ax0_ax1_fused_outer_outer & 1) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        T_matmul_NN_local[j_c_outer_inner] = (T_matmul_NN_local[j_c_outer_inner] + (data_shared[k_outer_inner] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 2)] = (T_matmul_NN_local[(j_c_outer_inner + 2)] + (data_shared[(k_outer_inner + 2)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 4)] = (T_matmul_NN_local[(j_c_outer_inner + 4)] + (data_shared[(k_outer_inner + 4)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 6)] = (T_matmul_NN_local[(j_c_outer_inner + 6)] + (data_shared[(k_outer_inner + 6)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 8)] = (T_matmul_NN_local[(j_c_outer_inner + 8)] + (data_shared[(k_outer_inner + 8)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 10)] = (T_matmul_NN_local[(j_c_outer_inner + 10)] + (data_shared[(k_outer_inner + 10)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 12)] = (T_matmul_NN_local[(j_c_outer_inner + 12)] + (data_shared[(k_outer_inner + 12)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 14)] = (T_matmul_NN_local[(j_c_outer_inner + 14)] + (data_shared[(k_outer_inner + 14)] * kernel_shared[(((k_outer_inner * 256) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
      }
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    T_matmul_NN[(((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 4096)] = T_matmul_NN_local[(j_inner + 2)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 8192)] = T_matmul_NN_local[(j_inner + 4)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 12288)] = T_matmul_NN_local[(j_inner + 6)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 16384)] = T_matmul_NN_local[(j_inner + 8)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 20480)] = T_matmul_NN_local[(j_inner + 10)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 24576)] = T_matmul_NN_local[(j_inner + 12)];
    T_matmul_NN[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)blockIdx.x) & 15) * 256)) + (((int)threadIdx.x) * 2)) + j_inner) + 28672)] = T_matmul_NN_local[(j_inner + 14)];
  }
}

