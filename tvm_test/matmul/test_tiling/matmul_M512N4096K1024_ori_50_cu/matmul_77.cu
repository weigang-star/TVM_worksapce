
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 64) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 64) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 128)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 64) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 256)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 64) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 384)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 64; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 64) {
        data_shared[((((int)threadIdx.x) * 64) + ax0_ax1_fused_inner_s)] = data[((((((((int)blockIdx.x) >> 3) * 262144) + (((int)threadIdx.x) * 4096)) + ((ax0_ax1_fused_inner_s >> 4) * 1024)) + (k_outer_outer * 16)) + (ax0_ax1_fused_inner_s & 15))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer & 1) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
            for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
              for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
                T_matmul_NN_local[((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 128)] = (T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 128)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] = (T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] + (data_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner) + 2048)] * kernel_shared[(((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 384)] = (T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 384)] + (data_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_c_outer_inner * 64)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner) + 2048)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)]));
              }
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 256)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 524544)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 384)];
    }
  }
}

