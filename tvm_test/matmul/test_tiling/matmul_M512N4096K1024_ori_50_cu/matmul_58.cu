
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 64; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[(((ax0_ax1_fused_outer_outer * 262144) + (((int)threadIdx.x) * 1024)) + k_outer_outer)];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 2; ++ax0_ax1_fused_outer_outer_1) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 1024) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + ((((k_outer_outer * 4096) + (((int)blockIdx.x) * 2048)) + (ax0_ax1_fused_outer_outer_1 * 1024)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 64; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 2)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + j_c_inner)]));
            T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 2)) + i_c_inner)] * kernel_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 128; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 6) * 524288) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 6) * 524288) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 2048)];
    }
  }
}

