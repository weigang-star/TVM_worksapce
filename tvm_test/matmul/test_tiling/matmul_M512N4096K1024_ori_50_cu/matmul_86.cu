
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 128) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 128) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 256)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 131072) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + ((ax0_ax1_fused_outer_outer_1 >> 2) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 3) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
            for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
              for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
                T_matmul_NN_local[((((i_c_outer_inner * 128) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 128) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
                T_matmul_NN_local[(((((i_c_outer_inner * 128) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] = (T_matmul_NN_local[(((((i_c_outer_inner * 128) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 256) + (i_c_outer_inner * 128)) + (i_c_inner * 16)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[((((((k_outer_inner * 4096) + (k_inner * 512)) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)]));
              }
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 524288) + ((((int)threadIdx.x) >> 4) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 524288) + ((((int)threadIdx.x) >> 4) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 256)];
    }
  }
}

