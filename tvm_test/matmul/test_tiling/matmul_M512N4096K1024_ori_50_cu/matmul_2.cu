
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[2048];
  T_matmul_NN_local[0] = 0.000000e+00f;
  T_matmul_NN_local[1] = 0.000000e+00f;
  T_matmul_NN_local[2] = 0.000000e+00f;
  T_matmul_NN_local[3] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(data_shared + ((ax0_ax1_fused_outer_outer * 1024) + (((int)threadIdx.x) * 4))) = *(float4*)(data + ((((((((int)blockIdx.x) >> 8) * 65536) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 128)) + ((((int)threadIdx.x) & 31) * 4)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 8; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 524288) + (ax0_ax1_fused_outer_outer_1 * 65536)) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 15))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 64; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[((((((int)threadIdx.x) >> 3) * 128) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((k_outer_inner * 32) + (k_inner * 16)) + (((int)threadIdx.x) & 7))]));
        T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[((((((int)threadIdx.x) >> 3) * 128) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 32) + (k_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        T_matmul_NN_local[2] = (T_matmul_NN_local[2] + (data_shared[(((((((int)threadIdx.x) >> 3) * 128) + (k_outer_inner * 2)) + k_inner) + 4096)] * kernel_shared[(((k_outer_inner * 32) + (k_inner * 16)) + (((int)threadIdx.x) & 7))]));
        T_matmul_NN_local[3] = (T_matmul_NN_local[3] + (data_shared[(((((((int)threadIdx.x) >> 3) * 128) + (k_outer_inner * 2)) + k_inner) + 4096)] * kernel_shared[((((k_outer_inner * 32) + (k_inner * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
      }
    }
  }
  T_matmul_NN[(((((((int)blockIdx.x) >> 8) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 7))] = T_matmul_NN_local[0];
  T_matmul_NN[((((((((int)blockIdx.x) >> 8) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 7)) + 8)] = T_matmul_NN_local[1];
  T_matmul_NN[((((((((int)blockIdx.x) >> 8) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 7)) + 131072)] = T_matmul_NN_local[2];
  T_matmul_NN[((((((((int)blockIdx.x) >> 8) * 262144) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 7)) + 131080)] = T_matmul_NN_local[3];
}

