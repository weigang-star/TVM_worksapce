
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[4096];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 32; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 32; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 8; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 256) + (j_c_outer_inner_init * 8)) + j_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 32; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 32) {
        data_shared[((((int)threadIdx.x) * 32) + ax0_ax1_fused_inner_s)] = data[((((((int)threadIdx.x) * 16384) + ((ax0_ax1_fused_inner_s >> 1) * 1024)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s & 1))];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 4) * 4096)) + (((int)blockIdx.x) * 2048)) + ((ax0_ax1_fused_outer_outer & 15) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 32; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 32; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 8; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_inner * 256) + (j_c_outer_inner * 8)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 256) + (j_c_outer_inner * 8)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 3) * 64) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 2048) + ((((int)threadIdx.x) & 7) * 256)) + (j_c_outer_inner * 8)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 256; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 7) * 256)) + j_inner)] = T_matmul_NN_local[((i_inner * 256) + j_inner)];
    }
  }
}

