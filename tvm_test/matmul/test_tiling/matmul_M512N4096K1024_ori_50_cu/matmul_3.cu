
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[64];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[512];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 16; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 2) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 32)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 4; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 128) {
        data_shared[((((int)threadIdx.x) * 4) + ax0_ax1_fused_inner_s)] = data[((((((((int)blockIdx.x) >> 4) * 262144) + (((int)threadIdx.x) * 2048)) + ((ax0_ax1_fused_inner_s >> 1) * 1024)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s & 1))];
      }
    }
    if (((int)threadIdx.x) < 128) {
      *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + ((((k_outer_outer * 8192) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 16; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
          T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 6) * 32) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 256) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 32)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 32)] + (data_shared[((((((int)threadIdx.x) >> 6) * 32) + (i_c_outer_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 1048576) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 4) * 1048576) + ((((int)threadIdx.x) >> 6) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 32)];
    }
  }
}

