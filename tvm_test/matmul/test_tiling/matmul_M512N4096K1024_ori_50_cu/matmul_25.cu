
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 64) + (i_c_inner_init * 4)) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 64) + (i_c_inner_init * 4)) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 256)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 131072) + (ax0_ax1_fused_outer_outer * 16384)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 32; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer_1 >> 2) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 3) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
            for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
              T_matmul_NN_local[((((i_c_outer_inner * 64) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 64) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[(((i_c_outer_inner * 128) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 512) + (((int)threadIdx.x) * 4)) + (j_c_outer_inner * 2)) + j_c_inner)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)] = (T_matmul_NN_local[(((((i_c_outer_inner * 64) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)] + (data_shared[((((i_c_outer_inner * 128) + (i_c_inner * 8)) + k_inner) + 512)] * kernel_shared[((((k_inner * 512) + (((int)threadIdx.x) * 4)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 64; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 4)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 256)];
    }
  }
}

