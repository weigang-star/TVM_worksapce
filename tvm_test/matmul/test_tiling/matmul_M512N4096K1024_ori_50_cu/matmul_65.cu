
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 128; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 128) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 128) + j_c_outer_inner_init) + 512)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(data_shared + ((ax0_ax1_fused_outer_outer * 32) + (((int)threadIdx.x) * 2))) = *(float2*)(data + ((((((((int)blockIdx.x) >> 5) * 131072) + (ax0_ax1_fused_outer_outer * 2048)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 128; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 16) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + ((ax0_ax1_fused_outer_outer_1 >> 3) * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((ax0_ax1_fused_outer_outer_1 & 7) * 16)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 128; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 8; ++k_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 128) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 128) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) * 64) + (i_c_outer_inner * 16)) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((k_outer_inner * 1024) + (k_inner * 128)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 128) + j_c_outer_inner) + 512)] = (T_matmul_NN_local[(((i_c_outer_inner * 128) + j_c_outer_inner) + 512)] + (data_shared[(((((((int)threadIdx.x) * 64) + (i_c_outer_inner * 16)) + (k_outer_inner * 8)) + k_inner) + 1024)] * kernel_shared[(((k_outer_inner * 1024) + (k_inner * 128)) + j_c_outer_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 5) * 524288) + (((int)threadIdx.x) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 5) * 524288) + (((int)threadIdx.x) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 512)];
    }
  }
}

