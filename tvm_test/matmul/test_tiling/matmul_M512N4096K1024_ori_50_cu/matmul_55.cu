
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 16; ++j_c_outer_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 64)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 128)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 192)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 256)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 320)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 384)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 448)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 262144) + (ax0_ax1_fused_outer_outer * 16384)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 32; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + ((ax0_ax1_fused_outer_outer_1 >> 1) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 1) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 16; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 16; ++k_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 128)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 128)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 128)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 192)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 192)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 384)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 256)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 256)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner) + 2048)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 320)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 320)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner) + 2048)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 128)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 384)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 384)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner) + 2048)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)]));
            T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 448)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (j_c_outer_inner * 2)) + j_c_inner) + 448)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + k_inner) + 2048)] * kernel_shared[(((((k_inner * 512) + ((((int)threadIdx.x) & 3) * 32)) + (j_c_outer_inner * 2)) + j_c_inner) + 384)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 32; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner)] = T_matmul_NN_local[((i_inner * 32) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 192)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 256)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 524416)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 320)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 524544)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 384)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 3) * 32)) + j_inner) + 524672)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 448)];
    }
  }
}

