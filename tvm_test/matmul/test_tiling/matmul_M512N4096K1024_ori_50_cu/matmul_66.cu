
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[2048];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 128; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 128) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 128) + j_c_outer_inner_init) + 2048)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 262144) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 32) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 4) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 15) * 32)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 128; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
          T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 128) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 1) * 64) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 512) + ((((int)threadIdx.x) & 1) * 128)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 2048)] = (T_matmul_NN_local[(((i_c_inner * 128) + j_c_outer_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 1) * 64) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 512) + ((((int)threadIdx.x) & 1) * 128)) + j_c_outer_inner) + 256)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 1) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 1) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 1048576) + ((((int)threadIdx.x) >> 1) * 65536)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 1) * 128)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 2048)];
    }
  }
}

