
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[8192];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 16; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 32; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 8; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 128) + (j_c_outer_inner_init * 8)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 128) + (j_c_outer_inner_init * 8)) + j_c_inner_init) + 4096)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 32768) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 4) * 4096)) + (((int)blockIdx.x) * 2048)) + ((ax0_ax1_fused_outer_outer_1 & 15) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 16; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 32; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 8; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_inner * 128) + (j_c_outer_inner * 8)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 128) + (j_c_outer_inner * 8)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 3) * 128) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 7) * 128)) + (j_c_outer_inner * 8)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 8)) + j_c_inner) + 4096)] = (T_matmul_NN_local[((((i_c_inner * 128) + (j_c_outer_inner * 8)) + j_c_inner) + 4096)] + (data_shared[((((((int)threadIdx.x) >> 3) * 128) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[(((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 7) * 128)) + (j_c_outer_inner * 8)) + j_c_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 128) + j_inner) + 4096)];
    }
  }
}

