
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[256];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 16; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 1024)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 256) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 3072)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[(((((int)blockIdx.x) * 262144) + (((int)threadIdx.x) * 1024)) + k_outer_outer)];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((k_outer_outer * 4096) + (ax0_ax1_fused_outer_outer * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 16; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
            T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 5) * 16) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 31) * 64) + (j_c_outer_inner * 4)) + j_c_inner)]));
            T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)] + (data_shared[((((((int)threadIdx.x) >> 5) * 16) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[(((((((int)threadIdx.x) & 31) * 64) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)]));
            T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 16) + (i_c_outer_inner * 4)) + i_c_inner) + 128)] * kernel_shared[((((((int)threadIdx.x) & 31) * 64) + (j_c_outer_inner * 4)) + j_c_inner)]));
            T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 3072)] = (T_matmul_NN_local[(((((i_c_outer_inner * 256) + (i_c_inner * 64)) + (j_c_outer_inner * 4)) + j_c_inner) + 3072)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 16) + (i_c_outer_inner * 4)) + i_c_inner) + 128)] * kernel_shared[(((((((int)threadIdx.x) & 31) * 64) + (j_c_outer_inner * 4)) + j_c_inner) + 2048)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 64; ++j_inner) {
      T_matmul_NN[(((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner)] = T_matmul_NN_local[((i_inner * 64) + j_inner)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 1024)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 2048)];
      T_matmul_NN[((((((((int)blockIdx.x) * 1048576) + ((((int)threadIdx.x) >> 5) * 65536)) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 526336)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 3072)];
    }
  }
}

