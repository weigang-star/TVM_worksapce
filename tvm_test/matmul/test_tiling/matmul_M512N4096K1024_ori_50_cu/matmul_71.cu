
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[256];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 8) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 96)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 128)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 160)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 192)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 8) + j_c_outer_inner_init) + 224)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 5) * 131072) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer_1 >> 1) * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((ax0_ax1_fused_outer_outer_1 & 1) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 4; ++k_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 8) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 8) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 32)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 32)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 128)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 64)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 64)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 256)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 96)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 96)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 384)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 128)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 128)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 512)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 160)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 160)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 640)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 192)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 192)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 768)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 224)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + j_c_outer_inner) + 224)] + (data_shared[((((((((int)threadIdx.x) >> 4) * 32) + (i_c_outer_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 896)] * kernel_shared[((((k_outer_inner * 512) + (k_inner * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_c_outer_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 65536)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 196608)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 96)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 327680)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 160)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 393216)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 192)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 5) * 524288) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + j_inner) + 458752)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 224)];
    }
  }
}

