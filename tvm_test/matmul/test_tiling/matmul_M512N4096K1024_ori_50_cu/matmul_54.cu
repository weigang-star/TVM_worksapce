
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 32; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 4) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_outer_inner_init) + 128)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_outer_inner_init) + 256)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_outer_inner_init) + 384)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 128; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 4096) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 65536) + (ax0_ax1_fused_outer_outer_1 * 4096)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 32; ++i_c_inner) {
            T_matmul_NN_local[((i_c_inner * 4) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 4) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 16)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 128)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 128)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 16)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
            T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 256)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 256)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 16)) + (k_outer_inner * 2)) + k_inner) + 4096)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 384)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_outer_inner) + 384)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 512) + (i_c_inner * 16)) + (k_outer_inner * 2)) + k_inner) + 4096)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_c_outer_inner) + 32)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 32)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 128)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 256)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 3) * 131072) + (i_inner * 4096)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + j_inner) + 1048608)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 384)];
    }
  }
}

