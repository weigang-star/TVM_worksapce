
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[64];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 2) + i_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + i_c_inner_init) + 4)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 512) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    if (((int)threadIdx.x) < 16) {
      *(float4*)(kernel_shared + (((int)threadIdx.x) * 4)) = *(float4*)(kernel + ((((k_outer_outer * 32768) + ((((int)threadIdx.x) >> 1) * 4096)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 1) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          T_matmul_NN_local[((i_c_outer_inner * 2) + i_c_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + i_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((k_outer_inner * 8) + (((int)threadIdx.x) & 7))]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + i_c_inner) + 4)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + i_c_inner) + 4)] + (data_shared[((((((((int)threadIdx.x) >> 3) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_outer_inner) + 2048)] * kernel_shared[((k_outer_inner * 8) + (((int)threadIdx.x) & 7))]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    T_matmul_NN[(((((((int)threadIdx.x) >> 3) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[((((((((int)threadIdx.x) >> 3) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 1048576)] = T_matmul_NN_local[(i_inner + 4)];
  }
}

