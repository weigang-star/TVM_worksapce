
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[8192];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 4) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 16)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 48)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 80)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 96)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 4) + j_c_inner_init) + 112)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + (((((k_outer_outer * 32768) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 8) * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 255) * 4)));
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
          T_matmul_NN_local[((i_c_inner * 4) + j_c_inner)] = (T_matmul_NN_local[((i_c_inner * 4) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[(((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 16)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 16)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 128)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 32)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 32)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 256)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 48)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 48)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 384)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 64)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 512)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 80)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 80)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 640)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 96)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 96)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 768)]));
          T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 112)] = (T_matmul_NN_local[(((i_c_inner * 4) + j_c_inner) + 112)] + (data_shared[((((((int)threadIdx.x) >> 5) * 32) + (i_c_inner * 8)) + k_inner)] * kernel_shared[((((k_inner * 1024) + ((((int)threadIdx.x) & 31) * 4)) + j_c_inner) + 896)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 32)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 48)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 64)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 640)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 80)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 768)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 96)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 5) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 31) * 4)) + j_inner) + 896)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 112)];
    }
  }
}

