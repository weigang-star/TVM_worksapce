
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 16) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 16) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 256)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 4; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 256) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 16384) + (ax0_ax1_fused_outer_outer_1 * 4096)) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_inner * 16) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 16) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 4) * 64) + (i_c_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 16) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] = (T_matmul_NN_local[((((i_c_inner * 16) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 64) + (i_c_inner * 4)) + k_outer_inner) + 1024)] * kernel_shared[((((k_outer_inner * 256) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 4) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 4) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 256)];
    }
  }
}

