
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[512];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 64; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 4096) + (i_c_inner_init * 64)) + (j_c_outer_inner_init * 32)) + j_c_inner_init)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 8) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 4) * 262144) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 8) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer_1 >> 5) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((ax0_ax1_fused_outer_outer_1 & 31) * 8)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 64; ++i_c_inner) {
            for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
              T_matmul_NN_local[((((i_c_outer_inner * 4096) + (i_c_inner * 64)) + (j_c_outer_inner * 32)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 4096) + (i_c_inner * 64)) + (j_c_outer_inner * 32)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 256) + (i_c_outer_inner * 128)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 256) + ((((int)threadIdx.x) & 3) * 64)) + (j_c_outer_inner * 32)) + j_c_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 128; ++i_inner) {
    for (int j_inner = 0; j_inner < 64; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 4) * 1048576) + ((((int)threadIdx.x) >> 2) * 524288)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 3) * 64)) + j_inner)] = T_matmul_NN_local[((i_inner * 64) + j_inner)];
    }
  }
}

