
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[64];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 32)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 64)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 96)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 16384) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((k_outer_outer * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 15))];
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int k_inner = 0; k_inner < 4; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 4)) + k_inner)] * kernel_shared[((k_inner * 16) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 32)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 32)] + (data_shared[((((((int)threadIdx.x) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 4)) + k_inner)] * kernel_shared[(((k_inner * 16) + j_c_inner) + 4)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 64)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 64)] + (data_shared[((((((int)threadIdx.x) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 4)) + k_inner)] * kernel_shared[(((k_inner * 16) + j_c_inner) + 8)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 96)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 96)] + (data_shared[((((((int)threadIdx.x) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 4)) + k_inner)] * kernel_shared[(((k_inner * 16) + j_c_inner) + 12)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((int)threadIdx.x) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[(((((((int)threadIdx.x) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + j_inner) + 4)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 32)];
      T_matmul_NN[(((((((int)threadIdx.x) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + j_inner) + 8)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 64)];
      T_matmul_NN[(((((((int)threadIdx.x) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + j_inner) + 12)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 96)];
    }
  }
}

