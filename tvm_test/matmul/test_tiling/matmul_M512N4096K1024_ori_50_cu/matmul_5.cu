
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[256];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    T_matmul_NN_local[j_c_outer_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 4)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 8) * 131072) + (ax0_ax1_fused_outer_outer * 16384)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[((((k_outer_outer * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + (((int)threadIdx.x) & 15))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 4; ++k_inner) {
          T_matmul_NN_local[j_c_outer_inner] = (T_matmul_NN_local[j_c_outer_inner] + (data_shared[((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 4)) + k_inner)] * kernel_shared[((((k_outer_inner * 64) + (k_inner * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
          T_matmul_NN_local[(j_c_outer_inner + 4)] = (T_matmul_NN_local[(j_c_outer_inner + 4)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 4)) + k_inner) + 1024)] * kernel_shared[((((k_outer_inner * 64) + (k_inner * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 4; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 8) * 524288) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 8) * 524288) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 255) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 262144)] = T_matmul_NN_local[(j_inner + 4)];
  }
}

