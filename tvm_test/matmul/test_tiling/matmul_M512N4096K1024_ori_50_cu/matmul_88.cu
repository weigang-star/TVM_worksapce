
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[128];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 8) + (i_c_inner_init * 4)) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 16)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 32768) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    if (((int)threadIdx.x) < 128) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((k_outer_outer * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 15))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
              T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + (j_c_outer_inner * 2)) + j_c_inner)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 8) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner) + 16)] = (T_matmul_NN_local[(((((i_c_outer_inner * 8) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner) + 16)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 16)) + (i_c_inner * 8)) + k_outer_inner) + 2048)] * kernel_shared[((((k_outer_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 2) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 2) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 16)];
    }
  }
}

