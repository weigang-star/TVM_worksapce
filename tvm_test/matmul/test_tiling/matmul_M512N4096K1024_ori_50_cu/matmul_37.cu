
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32768];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[8192];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 32; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 64; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 4096)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 8192)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 12288)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 16384)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 20480)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 24576)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_inner_init * 64) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 28672)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 32768) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 128; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((k_outer_outer * 8192) + (ax0_ax1_fused_outer_outer_1 * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 32; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 64; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 4096)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 4096)] + (data_shared[((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner)] * kernel_shared[(((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner) + 2048)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 8192)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 8192)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 256)] * kernel_shared[((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 12288)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 12288)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 256)] * kernel_shared[(((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner) + 2048)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 16384)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 16384)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 512)] * kernel_shared[((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 20480)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 20480)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 512)] * kernel_shared[(((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner) + 2048)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 24576)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 24576)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 768)] * kernel_shared[((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 28672)] = (T_matmul_NN_local[((((i_c_inner * 64) + (j_c_outer_inner * 2)) + j_c_inner) + 28672)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 128) + (i_c_inner * 2)) + k_outer_inner) + 768)] * kernel_shared[(((((k_outer_inner * 4096) + ((((int)threadIdx.x) & 31) * 64)) + (j_c_outer_inner * 2)) + j_c_inner) + 2048)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 64; ++i_inner) {
    for (int j_inner = 0; j_inner < 64; ++j_inner) {
      T_matmul_NN[(((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner)] = T_matmul_NN_local[((i_inner * 64) + j_inner)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 4096)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 8192)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 526336)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 12288)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 16384)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 1050624)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 20480)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 1572864)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 24576)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 5) * 262144) + (i_inner * 4096)) + ((((int)threadIdx.x) & 31) * 64)) + j_inner) + 1574912)] = T_matmul_NN_local[(((i_inner * 64) + j_inner) + 28672)];
    }
  }
}

