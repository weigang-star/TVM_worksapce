
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(4) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 64; ++i_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 16; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 16) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 1024)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 2048)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 3072)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 4096)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 5120)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 6144)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_inner_init) + 7168)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 1024; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(data_shared + ((ax0_ax1_fused_outer_outer * 8) + (((int)threadIdx.x) * 2))) = *(float2*)(data + (((((ax0_ax1_fused_outer_outer >> 1) * 1024) + (k_outer_outer * 16)) + ((ax0_ax1_fused_outer_outer & 1) * 8)) + (((int)threadIdx.x) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 256; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + ((ax0_ax1_fused_outer_outer_1 >> 4) * 4096)) + (((int)blockIdx.x) * 64)) + ((ax0_ax1_fused_outer_outer_1 & 15) * 4)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 64; ++i_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int j_c_inner = 0; j_c_inner < 16; ++j_c_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_inner)] + (data_shared[(((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 1024)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 1024)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 1024)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 2048)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 2048)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 2048)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 3072)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 3072)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 3072)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 4096)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 4096)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 4096)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 5120)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 5120)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 5120)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 6144)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 6144)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 6144)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 7168)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_inner) + 7168)] + (data_shared[((((i_c_outer_inner * 16) + (k_outer_inner * 2)) + k_inner) + 7168)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 64)) + (((int)threadIdx.x) * 16)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 64; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 1024)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 2048)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 786432)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 3072)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 4096)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 1310720)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 5120)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 1572864)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 6144)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 16)) + j_inner) + 1835008)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 7168)];
    }
  }
}

