
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 8; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 256) + (j_c_outer_inner_init * 32)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 4096)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 256) + (j_c_outer_inner_init * 32)) + j_c_inner_init) + 6144)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(data_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 2))) = *(float2*)(data + ((((((((int)blockIdx.x) >> 1) * 262144) + (ax0_ax1_fused_outer_outer * 32768)) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 128; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 5) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((ax0_ax1_fused_outer_outer_1 & 31) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
          for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 4)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 3) * 256)) + (j_c_outer_inner * 32)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 4)) + k_outer_inner)] * kernel_shared[(((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 3) * 256)) + (j_c_outer_inner * 32)) + j_c_inner) + 1024)]));
            T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 4096)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 4096)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 4)) + k_outer_inner) + 512)] * kernel_shared[((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 3) * 256)) + (j_c_outer_inner * 32)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 6144)] = (T_matmul_NN_local[((((i_c_outer_inner * 256) + (j_c_outer_inner * 32)) + j_c_inner) + 6144)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_outer_inner * 4)) + k_outer_inner) + 512)] * kernel_shared[(((((k_outer_inner * 2048) + ((((int)threadIdx.x) & 3) * 256)) + (j_c_outer_inner * 32)) + j_c_inner) + 1024)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 256; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 1048576) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 3) * 256)) + j_inner)] = T_matmul_NN_local[((i_inner * 256) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 1048576) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 3) * 256)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 256) + j_inner) + 2048)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 1048576) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 3) * 256)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 256) + j_inner) + 4096)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 1048576) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 3) * 256)) + j_inner) + 525312)] = T_matmul_NN_local[(((i_inner * 256) + j_inner) + 6144)];
    }
  }
}

