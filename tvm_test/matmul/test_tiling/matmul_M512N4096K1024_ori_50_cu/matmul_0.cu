
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[256];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    T_matmul_NN_local[i_c_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 4)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 8)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 12)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 16)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 20)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 24)] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 28)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 7) * 65536) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 64; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 4) {
        kernel_shared[((((int)threadIdx.x) * 64) + ax0_ax1_fused_inner_s)] = kernel[(((((k_outer_outer * 32768) + (((int)threadIdx.x) * 8192)) + ((ax0_ax1_fused_inner_s >> 5) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (ax0_ax1_fused_inner_s & 31))];
      }
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        T_matmul_NN_local[i_c_inner] = (T_matmul_NN_local[i_c_inner] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((k_outer_inner * 32) + (((int)threadIdx.x) & 3))]));
        T_matmul_NN_local[(i_c_inner + 4)] = (T_matmul_NN_local[(i_c_inner + 4)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 4)]));
        T_matmul_NN_local[(i_c_inner + 8)] = (T_matmul_NN_local[(i_c_inner + 8)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 8)]));
        T_matmul_NN_local[(i_c_inner + 12)] = (T_matmul_NN_local[(i_c_inner + 12)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 12)]));
        T_matmul_NN_local[(i_c_inner + 16)] = (T_matmul_NN_local[(i_c_inner + 16)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 16)]));
        T_matmul_NN_local[(i_c_inner + 20)] = (T_matmul_NN_local[(i_c_inner + 20)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 20)]));
        T_matmul_NN_local[(i_c_inner + 24)] = (T_matmul_NN_local[(i_c_inner + 24)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 24)]));
        T_matmul_NN_local[(i_c_inner + 28)] = (T_matmul_NN_local[(i_c_inner + 28)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 32) + (((int)threadIdx.x) & 3)) + 28)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 4)] = T_matmul_NN_local[(i_inner + 4)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 8)] = T_matmul_NN_local[(i_inner + 8)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 12)] = T_matmul_NN_local[(i_inner + 12)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 16)] = T_matmul_NN_local[(i_inner + 16)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 20)] = T_matmul_NN_local[(i_inner + 20)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 24)] = T_matmul_NN_local[(i_inner + 24)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 3)) + 28)] = T_matmul_NN_local[(i_inner + 28)];
  }
}

