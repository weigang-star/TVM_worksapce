
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[2048];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 32; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 512)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 1024)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + (j_c_outer_inner_init * 2)) + j_c_inner_init) + 1536)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 131072) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 32) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 4) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer_1 & 15) * 32)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 32; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 4; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
              T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[(((i_c_outer_inner * 8) + (i_c_inner * 4)) + k_inner)] * kernel_shared[((((k_inner * 512) + (((int)threadIdx.x) * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 512)] = (T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 512)] + (data_shared[(((i_c_outer_inner * 8) + (i_c_inner * 4)) + k_inner)] * kernel_shared[(((((k_inner * 512) + (((int)threadIdx.x) * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 1024)] = (T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 1024)] + (data_shared[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + k_inner) + 256)] * kernel_shared[((((k_inner * 512) + (((int)threadIdx.x) * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 1536)] = (T_matmul_NN_local[(((((i_c_outer_inner * 16) + (i_c_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 1536)] + (data_shared[((((i_c_outer_inner * 8) + (i_c_inner * 4)) + k_inner) + 256)] * kernel_shared[(((((k_inner * 512) + (((int)threadIdx.x) * 8)) + (j_c_outer_inner * 2)) + j_c_inner) + 256)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 64; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 8)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 512)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 8)) + j_inner) + 262144)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 1024)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 524288) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + (((int)threadIdx.x) * 8)) + j_inner) + 262400)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 1536)];
    }
  }
}

