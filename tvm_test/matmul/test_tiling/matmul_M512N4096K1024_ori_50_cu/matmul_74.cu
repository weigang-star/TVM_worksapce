
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(1024) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[4096];
  for (int i_c_inner_init = 0; i_c_inner_init < 16; ++i_c_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 32) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_inner_init * 32) + j_c_inner_init) + 512)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[((((((int)threadIdx.x) >> 1) * 1024) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 1024) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + (((int)blockIdx.x) * 2048)) + ((ax0_ax1_fused_outer_outer & 1) * 1024)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 16; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
          T_matmul_NN_local[((i_c_inner * 32) + j_c_inner)] = (T_matmul_NN_local[((i_c_inner * 32) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 6) * 32) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 2048) + ((((int)threadIdx.x) & 63) * 32)) + j_c_inner)]));
          T_matmul_NN_local[(((i_c_inner * 32) + j_c_inner) + 512)] = (T_matmul_NN_local[(((i_c_inner * 32) + j_c_inner) + 512)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 32) + (i_c_inner * 2)) + k_inner) + 512)] * kernel_shared[(((k_inner * 2048) + ((((int)threadIdx.x) & 63) * 32)) + j_c_inner)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 32; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 6) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 63) * 32)) + j_inner)] = T_matmul_NN_local[((i_inner * 32) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 6) * 65536) + (i_inner * 4096)) + (((int)blockIdx.x) * 2048)) + ((((int)threadIdx.x) & 63) * 32)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 512)];
    }
  }
}

