
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8192];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 512; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 2) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 1024)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 2048)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 3072)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 4096)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 5120)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 6144)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_outer_inner_init) + 7168)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 8192) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 128; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 32768) + ((ax0_ax1_fused_outer_outer_1 >> 4) * 4096)) + (((int)blockIdx.x) * 1024)) + ((ax0_ax1_fused_outer_outer_1 & 15) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 512; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_outer_inner)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[(((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 1024)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 1024)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 128)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 2048)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 2048)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 256)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 3072)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 3072)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 384)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 4096)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 4096)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 512)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 5120)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 5120)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 640)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 6144)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 6144)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 768)]));
          T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 7168)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_outer_inner) + 7168)] + (data_shared[((i_c_outer_inner * 8) + k_inner)] * kernel_shared[((((k_inner * 1024) + (((int)threadIdx.x) * 2)) + j_c_outer_inner) + 896)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 512; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 1024)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 2048)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 3072)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 4096)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 640)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 5120)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 768)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 6144)];
      T_matmul_NN[(((((i_inner * 4096) + (((int)blockIdx.x) * 1024)) + (((int)threadIdx.x) * 2)) + j_inner) + 896)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 7168)];
    }
  }
}

