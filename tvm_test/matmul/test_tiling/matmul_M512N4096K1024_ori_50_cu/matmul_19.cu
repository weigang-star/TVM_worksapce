
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[256];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 128)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 256)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 384)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 512)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 640)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 768)] = 0.000000e+00f;
          T_matmul_NN_local[(((((i_c_outer_inner_init * 32) + (i_c_inner_init * 16)) + (j_c_outer_inner_init * 4)) + j_c_inner_init) + 896)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 1) * 131072) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 3) * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((ax0_ax1_fused_outer_outer & 7) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
              T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 128)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 128)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 256)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 512)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 384)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 384)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 768)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 512)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 512)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 1024)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 640)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 640)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 1280)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 768)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 768)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 1536)]));
              T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 896)] = (T_matmul_NN_local[(((((i_c_outer_inner * 32) + (i_c_inner * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 896)] + (data_shared[(((((((int)threadIdx.x) >> 4) * 16) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((((k_inner * 2048) + ((((int)threadIdx.x) & 15) * 16)) + (j_c_outer_inner * 4)) + j_c_inner) + 1792)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 16; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner)] = T_matmul_NN_local[((i_inner * 16) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 128)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 256)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 768)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 384)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 512)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 1280)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 640)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 1536)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 768)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 1) * 524288) + ((((int)threadIdx.x) >> 4) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 1) * 2048)) + ((((int)threadIdx.x) & 15) * 16)) + j_inner) + 1792)] = T_matmul_NN_local[(((i_inner * 16) + j_inner) + 896)];
    }
  }
}

