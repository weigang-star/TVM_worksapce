
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[4096];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 32; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 512)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 1024)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 1536)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 2048)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 2560)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 3072)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + j_c_inner_init) + 3584)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    data_shared[((int)threadIdx.x)] = data[((((int)threadIdx.x) * 1024) + k_outer_outer)];
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + (((k_outer_outer * 4096) + (ax0_ax1_fused_outer_outer * 2048)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 32; ++i_c_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
          T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[(((((int)threadIdx.x) & 127) * 4) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 512)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 512)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 512)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 1024)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 1024)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 1024)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 1536)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 1536)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 1536)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 2048)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 2048)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 2048)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 2560)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 2560)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 2560)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 3072)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 3072)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 3072)]));
          T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 3584)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + j_c_inner) + 3584)] + (data_shared[((((((int)threadIdx.x) >> 7) * 128) + (i_c_outer_inner * 4)) + i_c_inner)] * kernel_shared[((((((int)threadIdx.x) & 127) * 4) + j_c_inner) + 3584)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 128; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 512)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 1024)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 1024)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 1536)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 1536)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 2048)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 2048)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 2560)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 2560)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 3072)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 3072)];
      T_matmul_NN[((((((((int)threadIdx.x) >> 7) * 524288) + (i_inner * 4096)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner) + 3584)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 3584)];
    }
  }
}

