
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[4096];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 32; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 128) + (j_c_outer_inner_init * 32)) + j_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 2) * 131072) + (ax0_ax1_fused_outer_outer * 65536)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 8; ++ax0_ax1_fused_outer_outer_1) {
      *(float2*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 512) + (((int)threadIdx.x) * 2))) = *(float2*)(kernel + (((((k_outer_outer * 16384) + ((ax0_ax1_fused_outer_outer_1 >> 1) * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((ax0_ax1_fused_outer_outer_1 & 1) * 512)) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 2; ++k_inner) {
            for (int j_c_inner = 0; j_c_inner < 32; ++j_c_inner) {
              T_matmul_NN_local[(((i_c_outer_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 128) + (j_c_outer_inner * 32)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 3) * 16) + (i_c_outer_inner * 4)) + (k_outer_inner * 2)) + k_inner)] * kernel_shared[(((((k_outer_inner * 2048) + (k_inner * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + (j_c_outer_inner * 32)) + j_c_inner)]));
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 128; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 524288) + ((((int)threadIdx.x) >> 3) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + j_inner)] = T_matmul_NN_local[((i_inner * 128) + j_inner)];
    }
  }
}

