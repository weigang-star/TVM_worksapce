
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[256];
  __shared__ float data_shared[64];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 32) + (i_c_inner_init * 4)) + j_c_inner_init) + 128)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 4; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 16) {
        data_shared[((((int)threadIdx.x) * 4) + ax0_ax1_fused_inner_s)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (ax0_ax1_fused_inner_s * 1024)) + k_outer_outer)];
      }
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 4096) + ((((int)blockIdx.x) & 3) * 1024)) + (ax0_ax1_fused_outer_outer * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
          T_matmul_NN_local[(((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_inner)] + (data_shared[((i_c_outer_inner * 8) + i_c_inner)] * kernel_shared[((((int)threadIdx.x) * 4) + j_c_inner)]));
          T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_inner) + 128)] = (T_matmul_NN_local[((((i_c_outer_inner * 32) + (i_c_inner * 4)) + j_c_inner) + 128)] + (data_shared[(((i_c_outer_inner * 8) + i_c_inner) + 32)] * kernel_shared[((((int)threadIdx.x) * 4) + j_c_inner)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 4)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 128)];
    }
  }
}

