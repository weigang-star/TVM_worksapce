
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(2) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[256];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 8; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
          T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 4)) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 512; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 2) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 9) * 32768) + ((ax0_ax1_fused_outer_outer >> 4) * 1024)) + (k_outer_outer * 32)) + ((ax0_ax1_fused_outer_outer & 15) * 2)) + ((int)threadIdx.x))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 32; ++ax0_ax1_fused_outer_outer_1) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 8) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + ((((k_outer_outer * 131072) + (ax0_ax1_fused_outer_outer_1 * 4096)) + ((((int)blockIdx.x) & 511) * 8)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 16; ++k_inner) {
            for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
              for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
                T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 4)) + (j_c_outer_inner * 2)) + j_c_inner)] + (data_shared[((((i_c_outer_inner * 128) + (i_c_inner * 32)) + (k_outer_inner * 16)) + k_inner)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 8)) + (((int)threadIdx.x) * 4)) + (j_c_outer_inner * 2)) + j_c_inner)]));
              }
            }
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 32; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 9) * 131072) + (i_inner * 4096)) + ((((int)blockIdx.x) & 511) * 8)) + (((int)threadIdx.x) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
    }
  }
}

