
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[64];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 8) + (i_c_inner_init * 2)) + j_c_inner_init) + 16)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 16384) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    if (((int)threadIdx.x) < 64) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((k_outer_outer * 32768) + ((((int)threadIdx.x) >> 3) * 4096)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 64) + (i_c_outer_inner * 32)) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 16)] = (T_matmul_NN_local[((((i_c_outer_inner * 8) + (i_c_inner * 2)) + j_c_inner) + 16)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 64) + (i_c_outer_inner * 32)) + (i_c_inner * 8)) + k_outer_inner) + 2048)] * kernel_shared[(((k_outer_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[((((((((int)threadIdx.x) >> 2) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[(((((((((int)threadIdx.x) >> 2) * 32768) + (i_inner * 4096)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
    }
  }
}

