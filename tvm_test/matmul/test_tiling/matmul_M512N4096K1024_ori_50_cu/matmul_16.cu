
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[8];
  __shared__ float data_shared[512];
  __shared__ float kernel_shared[64];
  for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
    T_matmul_NN_local[i_c_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(i_c_inner_init + 4)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(data_shared + ((ax0_ax1_fused_outer_outer * 256) + (((int)threadIdx.x) * 4))) = *(float4*)(data + ((((((((int)blockIdx.x) >> 9) * 65536) + (ax0_ax1_fused_outer_outer * 32768)) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)));
    }
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 64; ++ax0_ax1_fused_inner_s) {
      if (((int)threadIdx.x) < 1) {
        kernel_shared[((((int)threadIdx.x) * 64) + ax0_ax1_fused_inner_s)] = kernel[(((((((int)threadIdx.x) * 32768) + (k_outer_outer * 32768)) + ((ax0_ax1_fused_inner_s >> 3) * 4096)) + ((((int)blockIdx.x) & 511) * 8)) + (ax0_ax1_fused_inner_s & 7))];
      }
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        T_matmul_NN_local[i_c_inner] = (T_matmul_NN_local[i_c_inner] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[((k_outer_inner * 8) + (((int)threadIdx.x) & 3))]));
        T_matmul_NN_local[(i_c_inner + 4)] = (T_matmul_NN_local[(i_c_inner + 4)] + (data_shared[((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 8)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 8) + (((int)threadIdx.x) & 3)) + 4)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 9) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 511) * 8)) + (((int)threadIdx.x) & 3))] = T_matmul_NN_local[i_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 9) * 262144) + ((((int)threadIdx.x) >> 2) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 511) * 8)) + (((int)threadIdx.x) & 3)) + 4)] = T_matmul_NN_local[(i_inner + 4)];
  }
}

