
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 8)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 16)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 4) + (i_c_inner_init * 2)) + j_c_outer_inner_init) + 24)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 3) * 16384) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 256) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 1) * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((ax0_ax1_fused_outer_outer & 1) * 256)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 2; ++k_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 8) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 8)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 8)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 8) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 128)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 16)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 16)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 8) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 256)]));
            T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 24)] = (T_matmul_NN_local[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + j_c_outer_inner) + 24)] + (data_shared[(((((((int)threadIdx.x) >> 6) * 8) + (i_c_outer_inner * 4)) + (i_c_inner * 2)) + k_inner)] * kernel_shared[((((k_inner * 512) + ((((int)threadIdx.x) & 63) * 2)) + j_c_outer_inner) + 384)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 128)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 8)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 256)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 2)) + j_inner) + 384)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 24)];
    }
  }
}

