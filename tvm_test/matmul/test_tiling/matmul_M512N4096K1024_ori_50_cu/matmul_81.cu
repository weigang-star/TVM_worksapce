
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[16];
  __shared__ float data_shared[1024];
  __shared__ float kernel_shared[512];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 4; ++j_c_inner_init) {
      T_matmul_NN_local[((j_c_outer_inner_init * 4) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 4) + j_c_inner_init) + 8)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 7) * 65536) + (ax0_ax1_fused_outer_outer * 8192)) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 4; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 65536) + (ax0_ax1_fused_outer_outer_1 * 16384)) + ((((int)threadIdx.x) >> 5) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + (((int)threadIdx.x) & 31))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          for (int j_c_inner = 0; j_c_inner < 4; ++j_c_inner) {
            T_matmul_NN_local[((j_c_outer_inner * 4) + j_c_inner)] = (T_matmul_NN_local[((j_c_outer_inner * 4) + j_c_inner)] + (data_shared[((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 8)) + k_inner)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 8)) + (j_c_outer_inner * 4)) + j_c_inner)]));
            T_matmul_NN_local[(((j_c_outer_inner * 4) + j_c_inner) + 8)] = (T_matmul_NN_local[(((j_c_outer_inner * 4) + j_c_inner) + 8)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 16) + (k_outer_inner * 8)) + k_inner) + 512)] * kernel_shared[(((((k_outer_inner * 256) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 8)) + (j_c_outer_inner * 4)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 8; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + j_inner) + 131072)] = T_matmul_NN_local[(j_inner + 8)];
  }
}

