
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[128];
  __shared__ float data_shared[2048];
  __shared__ float kernel_shared[2048];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 2; ++j_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 16; ++j_c_inner_init) {
      T_matmul_NN_local[((j_c_outer_inner_init * 16) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 16) + j_c_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 16) + j_c_inner_init) + 64)] = 0.000000e+00f;
      T_matmul_NN_local[(((j_c_outer_inner_init * 16) + j_c_inner_init) + 96)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[(((((((int)blockIdx.x) >> 6) * 65536) + (ax0_ax1_fused_outer_outer * 1024)) + (k_outer_outer * 32)) + ((int)threadIdx.x))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      *(float4*)(kernel_shared + ((ax0_ax1_fused_outer_outer_1 * 128) + (((int)threadIdx.x) * 4))) = *(float4*)(kernel + (((((k_outer_outer * 131072) + (ax0_ax1_fused_outer_outer_1 * 8192)) + ((((int)threadIdx.x) >> 4) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 2; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 32; ++k_inner) {
        for (int j_c_inner = 0; j_c_inner < 16; ++j_c_inner) {
          T_matmul_NN_local[((j_c_outer_inner * 16) + j_c_inner)] = (T_matmul_NN_local[((j_c_outer_inner * 16) + j_c_inner)] + (data_shared[(((((int)threadIdx.x) >> 1) * 32) + k_inner)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + (j_c_outer_inner * 16)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 32)] = (T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 32)] + (data_shared[((((((int)threadIdx.x) >> 1) * 32) + k_inner) + 512)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + (j_c_outer_inner * 16)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 64)] = (T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 64)] + (data_shared[((((((int)threadIdx.x) >> 1) * 32) + k_inner) + 1024)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + (j_c_outer_inner * 16)) + j_c_inner)]));
          T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 96)] = (T_matmul_NN_local[(((j_c_outer_inner * 16) + j_c_inner) + 96)] + (data_shared[((((((int)threadIdx.x) >> 1) * 32) + k_inner) + 1536)] * kernel_shared[((((k_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + (j_c_outer_inner * 16)) + j_c_inner)]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 32; ++j_inner) {
    T_matmul_NN[((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + j_inner)] = T_matmul_NN_local[j_inner];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + j_inner) + 65536)] = T_matmul_NN_local[(j_inner + 32)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + j_inner) + 131072)] = T_matmul_NN_local[(j_inner + 64)];
    T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + j_inner) + 196608)] = T_matmul_NN_local[(j_inner + 96)];
  }
}

