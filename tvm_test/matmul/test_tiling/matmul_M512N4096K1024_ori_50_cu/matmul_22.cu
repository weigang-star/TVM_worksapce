
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[32];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[8192];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 8; ++i_c_outer_inner_init) {
    for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 2) + j_c_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 2) + j_c_inner_init) + 16)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 64; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 6) * 32768) + ((ax0_ax1_fused_outer_outer >> 1) * 1024)) + (k_outer_outer * 128)) + ((ax0_ax1_fused_outer_outer & 1) * 64)) + ((int)threadIdx.x))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 128; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 64) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 524288) + (ax0_ax1_fused_outer_outer_1 * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
        for (int k_inner = 0; k_inner < 16; ++k_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 2) + j_c_inner)] + (data_shared[(((((((int)threadIdx.x) >> 5) * 1024) + (i_c_outer_inner * 128)) + (k_outer_inner * 16)) + k_inner)] * kernel_shared[((((k_outer_inner * 1024) + (k_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + j_c_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_inner) + 16)] = (T_matmul_NN_local[(((i_c_outer_inner * 2) + j_c_inner) + 16)] + (data_shared[((((((((int)threadIdx.x) >> 5) * 1024) + (i_c_outer_inner * 128)) + (k_outer_inner * 16)) + k_inner) + 2048)] * kernel_shared[((((k_outer_inner * 1024) + (k_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 5) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + j_inner)] = T_matmul_NN_local[((i_inner * 2) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 6) * 131072) + ((((int)threadIdx.x) >> 5) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 63) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + j_inner) + 65536)] = T_matmul_NN_local[(((i_inner * 2) + j_inner) + 16)];
    }
  }
}

