
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[16];
  __shared__ float data_shared[32];
  __shared__ float kernel_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      T_matmul_NN_local[((i_c_inner_init * 4) + j_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      data_shared[((int)threadIdx.x)] = data[(((((((int)blockIdx.x) >> 3) * 16384) + ((((int)threadIdx.x) >> 1) * 1024)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 512) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 8192) + (ax0_ax1_fused_outer_outer * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
          T_matmul_NN_local[((i_c_inner * 4) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_inner * 4) + j_c_outer_inner)] + (data_shared[((((((int)threadIdx.x) >> 7) * 8) + (i_c_inner * 2)) + k_inner)] * kernel_shared[(((k_inner * 512) + ((((int)threadIdx.x) & 127) * 4)) + j_c_outer_inner)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 7) * 16384)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 127) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
    }
  }
}

