
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[64];
  __shared__ float data_shared[4096];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      T_matmul_NN_local[((i_c_outer_inner_init * 16) + j_c_outer_inner_init)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 32)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 4)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 36)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 8)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 40)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 12)] = 0.000000e+00f;
      T_matmul_NN_local[(((i_c_outer_inner_init * 16) + j_c_outer_inner_init) + 44)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 128; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 7) * 65536) + ((ax0_ax1_fused_outer_outer >> 1) * 1024)) + (k_outer_outer * 64)) + ((ax0_ax1_fused_outer_outer & 1) * 32)) + ((int)threadIdx.x))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 64; ++ax0_ax1_fused_outer_outer_1) {
      kernel_shared[((ax0_ax1_fused_outer_outer_1 * 32) + ((int)threadIdx.x))] = kernel[((((k_outer_outer * 262144) + (ax0_ax1_fused_outer_outer_1 * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
          for (int k_inner = 0; k_inner < 4; ++k_inner) {
            T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_outer_inner)] = (T_matmul_NN_local[((i_c_outer_inner * 16) + j_c_outer_inner)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 32)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 32)] + (data_shared[(((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 16)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 4)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 4)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 64)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 36)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 36)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 64)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 16)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 8)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 8)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 128)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 40)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 40)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 128)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 16)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 12)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 12)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 192)] * kernel_shared[((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner)]));
            T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 44)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + j_c_outer_inner) + 44)] + (data_shared[((((((((int)threadIdx.x) >> 2) * 512) + (i_c_outer_inner * 256)) + (k_outer_inner * 4)) + k_inner) + 192)] * kernel_shared[(((((k_outer_inner * 128) + (k_inner * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_c_outer_inner) + 16)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      T_matmul_NN[(((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner)] = T_matmul_NN_local[((i_inner * 4) + j_inner)];
      T_matmul_NN[((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 2) * 32768)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 16)] = T_matmul_NN_local[(((i_inner * 4) + j_inner) + 32)];
    }
  }
}

