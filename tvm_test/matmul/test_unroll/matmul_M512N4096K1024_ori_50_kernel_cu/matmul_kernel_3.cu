#include "hip/hip_runtime.h"
#define REPEAT 928
dim3 dimGrid(128, 1, 1);
dim3 dimBlock(32, 1, 1);
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN);

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "nvmlPower.hpp"

#define M 512
#define N 4096
#define K 1024







int main() {
    int repeat = REPEAT;
    float warmupRatio = 0.1;
    float randMax = 1000.0;
    srand((unsigned)time(NULL));
    
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * M * K);
    b   = (float*)malloc(sizeof(float) * N * K);
    out = (float*)malloc(sizeof(float) * M * N);

    // Initialize host arrays
    for(int i = 0; i < M * K; i++) {
        a[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }
    for(int i = 0; i < N * K; i++) {
        b[i] = (float)(rand() / (float)RAND_MAX - 0.5) * 2.0 * randMax;
    }

    // Allocate device memory 
    hipMalloc((void**)&d_a,   sizeof(float) * M * K);
    hipMalloc((void**)&d_b,   sizeof(float) * N * K);
    hipMalloc((void**)&d_out, sizeof(float) * M * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * K, hipMemcpyHostToDevice);

    // Executing kernel 
    for(int i = 0; i < repeat * warmupRatio; i++) {
        mymatmul_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    }
    hipDeviceSynchronize();
    nvmlAPIRun();
    for(int i = 0; i < repeat; i++) {
        mymatmul_kernel0<<<dimGrid, dimBlock>>>(d_a, d_b, d_out);
    }
    hipDeviceSynchronize();
    nvmlAPIEnd();
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N * M, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
