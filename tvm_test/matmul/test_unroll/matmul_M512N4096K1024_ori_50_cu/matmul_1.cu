
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[512];
  __shared__ float data_shared[8192];
  __shared__ float kernel_shared[512];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 32; ++j_c_outer_inner_init) {
    T_matmul_NN_local[j_c_outer_inner_init] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 128)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 256)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 384)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 32)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 160)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 288)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 416)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 64)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 192)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 320)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 448)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 96)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 224)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 352)] = 0.000000e+00f;
    T_matmul_NN_local[(j_c_outer_inner_init + 480)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 256; ++ax0_ax1_fused_outer_outer) {
      data_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = data[((((ax0_ax1_fused_outer_outer * 2048) + ((((int)threadIdx.x) >> 4) * 1024)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    kernel_shared[((int)threadIdx.x)] = kernel[(((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x))];
    kernel_shared[(((int)threadIdx.x) + 32)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 4096)];
    kernel_shared[(((int)threadIdx.x) + 64)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 8192)];
    kernel_shared[(((int)threadIdx.x) + 96)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 12288)];
    kernel_shared[(((int)threadIdx.x) + 128)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 16384)];
    kernel_shared[(((int)threadIdx.x) + 160)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 20480)];
    kernel_shared[(((int)threadIdx.x) + 192)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 24576)];
    kernel_shared[(((int)threadIdx.x) + 224)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 28672)];
    kernel_shared[(((int)threadIdx.x) + 256)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 32768)];
    kernel_shared[(((int)threadIdx.x) + 288)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 36864)];
    kernel_shared[(((int)threadIdx.x) + 320)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 40960)];
    kernel_shared[(((int)threadIdx.x) + 352)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 45056)];
    kernel_shared[(((int)threadIdx.x) + 384)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 49152)];
    kernel_shared[(((int)threadIdx.x) + 416)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 53248)];
    kernel_shared[(((int)threadIdx.x) + 448)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 57344)];
    kernel_shared[(((int)threadIdx.x) + 480)] = kernel[((((k_outer_outer * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 61440)];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 32; ++j_c_outer_inner) {
        T_matmul_NN_local[j_c_outer_inner] = (T_matmul_NN_local[j_c_outer_inner] + (data_shared[((((int)threadIdx.x) * 64) + k_outer_inner)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 128)] = (T_matmul_NN_local[(j_c_outer_inner + 128)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 2048)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 256)] = (T_matmul_NN_local[(j_c_outer_inner + 256)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 4096)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 384)] = (T_matmul_NN_local[(j_c_outer_inner + 384)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 6144)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 32)] = (T_matmul_NN_local[(j_c_outer_inner + 32)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 16)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 160)] = (T_matmul_NN_local[(j_c_outer_inner + 160)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 2064)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 288)] = (T_matmul_NN_local[(j_c_outer_inner + 288)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 4112)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 416)] = (T_matmul_NN_local[(j_c_outer_inner + 416)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 6160)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 64)] = (T_matmul_NN_local[(j_c_outer_inner + 64)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 32)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 192)] = (T_matmul_NN_local[(j_c_outer_inner + 192)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 2080)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 320)] = (T_matmul_NN_local[(j_c_outer_inner + 320)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 4128)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 448)] = (T_matmul_NN_local[(j_c_outer_inner + 448)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 6176)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 96)] = (T_matmul_NN_local[(j_c_outer_inner + 96)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 48)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 224)] = (T_matmul_NN_local[(j_c_outer_inner + 224)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 2096)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 352)] = (T_matmul_NN_local[(j_c_outer_inner + 352)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 4144)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
        T_matmul_NN_local[(j_c_outer_inner + 480)] = (T_matmul_NN_local[(j_c_outer_inner + 480)] + (data_shared[(((((int)threadIdx.x) * 64) + k_outer_inner) + 6192)] * kernel_shared[((k_outer_inner * 32) + j_c_outer_inner)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 32; ++j_inner) {
      T_matmul_NN[((((((int)threadIdx.x) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 32)) + j_inner)] = T_matmul_NN_local[((i_inner * 32) + j_inner)];
      T_matmul_NN[(((((((int)threadIdx.x) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 32)) + j_inner) + 524288)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 128)];
      T_matmul_NN[(((((((int)threadIdx.x) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 32)) + j_inner) + 1048576)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 256)];
      T_matmul_NN[(((((((int)threadIdx.x) * 16384) + (i_inner * 4096)) + (((int)blockIdx.x) * 32)) + j_inner) + 1572864)] = T_matmul_NN_local[(((i_inner * 32) + j_inner) + 384)];
    }
  }
}

