
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymatmul_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[1024];
  __shared__ float data_shared[128];
  __shared__ float kernel_shared[2048];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 8; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 8; ++j_c_inner_init) {
        T_matmul_NN_local[(((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 128)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 256)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 384)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 512)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 640)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 768)] = 0.000000e+00f;
        T_matmul_NN_local[((((i_c_outer_inner_init * 16) + (i_c_inner_init * 8)) + j_c_inner_init) + 896)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 512; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 16) {
      data_shared[(((int)threadIdx.x) * 8)] = data[((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2))];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 1)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 1)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 2)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 1024)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 3)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 1025)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 4)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 2048)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 5)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 2049)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 6)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 3072)];
    }
    if (((int)threadIdx.x) < 16) {
      data_shared[((((int)threadIdx.x) * 8) + 7)] = data[(((((((int)blockIdx.x) >> 2) * 65536) + (((int)threadIdx.x) * 4096)) + (k_outer_outer * 2)) + 3073)];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 64) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 8192) + ((ax0_ax1_fused_outer_outer >> 4) * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((ax0_ax1_fused_outer_outer & 15) * 64)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 8; ++j_c_inner) {
            T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner)] = (T_matmul_NN_local[(((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner)] + (data_shared[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner)] * kernel_shared[(((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 128)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 128)] + (data_shared[(((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner)] * kernel_shared[((((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner) + 512)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 256)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 256)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 32)] * kernel_shared[(((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 384)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 384)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 32)] * kernel_shared[((((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner) + 512)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 512)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 512)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 64)] * kernel_shared[(((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 640)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 640)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 64)] * kernel_shared[((((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner) + 512)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 768)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 768)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 96)] * kernel_shared[(((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner)]));
            T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 896)] = (T_matmul_NN_local[((((i_c_outer_inner * 16) + (i_c_inner * 8)) + j_c_inner) + 896)] + (data_shared[((((i_c_outer_inner * 4) + (i_c_inner * 2)) + k_outer_inner) + 96)] * kernel_shared[((((k_outer_inner * 1024) + (((int)threadIdx.x) * 8)) + j_c_inner) + 512)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_matmul_NN[((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner)] = T_matmul_NN_local[((i_inner * 8) + j_inner)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 512)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 128)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 65536)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 256)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 66048)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 384)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 131072)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 512)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 131584)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 640)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 196608)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 768)];
      T_matmul_NN[(((((((((int)blockIdx.x) >> 2) * 262144) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + (((int)threadIdx.x) * 8)) + j_inner) + 197120)] = T_matmul_NN_local[(((i_inner * 8) + j_inner) + 896)];
    }
  }
}

