
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) mymv_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[2];
  __shared__ float data_shared[8];
  __shared__ float kernel_shared[1024];
  T_matmul_NN_local[0] = 0.000000e+00f;
  T_matmul_NN_local[1] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 8) {
      data_shared[((int)threadIdx.x)] = data[((k_outer_outer * 8) + ((int)threadIdx.x))];
    }
    *(float2*)(kernel_shared + (((int)threadIdx.x) * 2)) = *(float2*)(kernel + (((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 128)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 32768));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 65536));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 384)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 98304));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 131072));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 640)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 163840));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 196608));
    *(float2*)(kernel_shared + ((((int)threadIdx.x) * 2) + 896)) = *(float2*)(kernel + ((((k_outer_outer * 262144) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 2)) + 229376));
    __syncthreads();
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[0] * kernel_shared[((int)threadIdx.x)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[0] * kernel_shared[(((int)threadIdx.x) + 64)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[1] * kernel_shared[(((int)threadIdx.x) + 128)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[1] * kernel_shared[(((int)threadIdx.x) + 192)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[2] * kernel_shared[(((int)threadIdx.x) + 256)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[2] * kernel_shared[(((int)threadIdx.x) + 320)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[3] * kernel_shared[(((int)threadIdx.x) + 384)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[3] * kernel_shared[(((int)threadIdx.x) + 448)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[4] * kernel_shared[(((int)threadIdx.x) + 512)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[4] * kernel_shared[(((int)threadIdx.x) + 576)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[5] * kernel_shared[(((int)threadIdx.x) + 640)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[5] * kernel_shared[(((int)threadIdx.x) + 704)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[6] * kernel_shared[(((int)threadIdx.x) + 768)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[6] * kernel_shared[(((int)threadIdx.x) + 832)]));
    T_matmul_NN_local[0] = (T_matmul_NN_local[0] + (data_shared[7] * kernel_shared[(((int)threadIdx.x) + 896)]));
    T_matmul_NN_local[1] = (T_matmul_NN_local[1] + (data_shared[7] * kernel_shared[(((int)threadIdx.x) + 960)]));
  }
  T_matmul_NN[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = T_matmul_NN_local[0];
  T_matmul_NN[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 64)] = T_matmul_NN_local[1];
}

