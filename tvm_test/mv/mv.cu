
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) mymv_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ T_matmul_NN) {
  float T_matmul_NN_local[24];
  __shared__ float data_shared[3];
  __shared__ float kernel_shared[9216];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 3; ++j_c_outer_inner_init) {
    T_matmul_NN_local[(j_c_outer_inner_init * 8)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 1)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 2)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 3)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 4)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 5)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 6)] = 0.000000e+00f;
    T_matmul_NN_local[((j_c_outer_inner_init * 8) + 7)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4096; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 1) {
      *(float3*)(data_shared + (((int)threadIdx.x) * 3)) = *(float3*)(data + ((((int)threadIdx.x) * 12288) + (k_outer_outer * 3)));
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 72; ++ax0_ax1_fused_outer_outer) {
      kernel_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = kernel[(((((k_outer_outer * 147456) + ((ax0_ax1_fused_outer_outer / 24) * 49152)) + (((int)blockIdx.x) * 3072)) + ((ax0_ax1_fused_outer_outer % 24) * 128)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 3; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 3; ++j_c_outer_inner) {
        T_matmul_NN_local[(j_c_outer_inner * 8)] = (T_matmul_NN_local[(j_c_outer_inner * 8)] + (data_shared[k_outer_inner] * kernel_shared[(((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8))]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 1)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 1)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 1)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 2)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 2)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 2)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 3)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 3)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 3)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 4)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 4)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 4)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 5)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 5)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 5)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 6)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 6)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 6)]));
        T_matmul_NN_local[((j_c_outer_inner * 8) + 7)] = (T_matmul_NN_local[((j_c_outer_inner * 8) + 7)] + (data_shared[k_outer_inner] * kernel_shared[((((k_outer_inner * 3072) + (((int)threadIdx.x) * 24)) + (j_c_outer_inner * 8)) + 7)]));
      }
    }
  }
  for (int j_inner = 0; j_inner < 24; ++j_inner) {
    T_matmul_NN[(((((int)blockIdx.x) * 3072) + (((int)threadIdx.x) * 24)) + j_inner)] = T_matmul_NN_local[j_inner];
  }
}

