
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) mydwconv_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[16];
  __shared__ float PaddedInput_shared[896];
  __shared__ float kernel_shared[8];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    for (int c_c_inner_init = 0; c_c_inner_init < 4; ++c_c_inner_init) {
      DepthwiseConv2d_local[((b_c_inner_init * 4) + c_c_inner_init)] = 0.000000e+00f;
      DepthwiseConv2d_local[(((b_c_inner_init * 4) + c_c_inner_init) + 8)] = 0.000000e+00f;
    }
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        PaddedInput_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x))] = (((((1 <= (di_outer_outer + (((int)blockIdx.x) % 28))) && ((di_outer_outer + (((int)blockIdx.x) % 28)) < 29)) && (1 <= (dj_outer_outer + (((int)threadIdx.x) % 28)))) && ((dj_outer_outer + (((int)threadIdx.x) % 28)) < 29)) ? data[(((((((((((((int)blockIdx.x) / 448) * 401408) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 2) * 100352)) + (((((int)blockIdx.x) % 448) / 28) * 6272)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 3) * 1568)) + ((((int)threadIdx.x) / 28) * 784)) + (di_outer_outer * 28)) + ((((int)blockIdx.x) % 28) * 28)) + dj_outer_outer) + (((int)threadIdx.x) % 28)) - 29)] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 8) {
        kernel_shared[((int)threadIdx.x)] = kernel[((((((((int)blockIdx.x) % 448) / 28) * 72) + (((int)threadIdx.x) * 9)) + (di_outer_outer * 3)) + dj_outer_outer)];
      }
      __syncthreads();
      for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
        for (int c_c_inner = 0; c_c_inner < 4; ++c_c_inner) {
          DepthwiseConv2d_local[((b_c_inner * 4) + c_c_inner)] = (DepthwiseConv2d_local[((b_c_inner * 4) + c_c_inner)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 448) + (b_c_inner * 224)) + (c_c_inner * 28)) + (((int)threadIdx.x) % 28))] * kernel_shared[c_c_inner]));
          DepthwiseConv2d_local[(((b_c_inner * 4) + c_c_inner) + 8)] = (DepthwiseConv2d_local[(((b_c_inner * 4) + c_c_inner) + 8)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 448) + (b_c_inner * 224)) + (c_c_inner * 28)) + (((int)threadIdx.x) % 28)) + 112)] * kernel_shared[(c_c_inner + 4)]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int c_inner = 0; c_inner < 4; ++c_inner) {
      DepthwiseConv2d[((((((((((int)blockIdx.x) / 448) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 28) * 6272)) + (c_inner * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 28))] = DepthwiseConv2d_local[((b_inner * 4) + c_inner)];
      DepthwiseConv2d[(((((((((((int)blockIdx.x) / 448) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (b_inner * 100352)) + (((((int)blockIdx.x) % 448) / 28) * 6272)) + (c_inner * 784)) + ((((int)blockIdx.x) % 28) * 28)) + (((int)threadIdx.x) % 28)) + 3136)] = DepthwiseConv2d_local[(((b_inner * 4) + c_inner) + 8)];
    }
  }
}

